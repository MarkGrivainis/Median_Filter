#include "hip/hip_runtime.h"
#ifndef __HIPCC__  
#define __HIPCC__
#endif


#define BLOCK_DIM 4
#define T 16

//for the kernels that uses fixed sizes
#define N 4096
#define M 4096
#define BLOCK_W 32
#define BLOCK_H 32
#define R 10
#define D (R*2+1)
#define D2 D*D
#define TILE_W (BLOCK_W - (2*R))
#define TILE_H (BLOCK_H - (2*R))

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <string>

#include "Grid.h"
#include "ReadWrite.h"
#include "Filter.h"
//texture<int, hipTextureType1D, hipReadModeElementType> tex;
hipError_t median_filter(Grid &output, const Grid &input, const int filter_radius, int padding_type);

//__global__ void median_Kernel_border_padding(int *inputImage, int *outputImage, int width, int height)//int *inputImage, int *outputImage, unsigned int width, unsigned int height)
//{
//	__shared__ int smem[(BLOCK_DIM + 2 * R)][(BLOCK_DIM + 2 * R)];
//	int x_s = threadIdx.x + R;
//	int y_s = threadIdx.y + R;
//
//	int x = blockIdx.x*blockDim.x + threadIdx.x;
//	int y = blockIdx.y*blockDim.y + threadIdx.y;
//	if (x < N && y < N)
//	{
//		smem[y_s][x_s] = inputImage[y*(N)+x];
//
//		if (threadIdx.x == 0)
//		{
//			if (threadIdx.y == 0)
//			{
//				for (int j = -1; j >= -R; --j)
//					for (int i = -1; i >= -R; --i)
//					{
//						int offset_x = max(0, x + i);
//						int offset_y = max(0, y + j);
//						smem[y_s + j][x_s + i] = inputImage[offset_y*(N)+offset_x];
//					}
//
//			}
//			if (threadIdx.y == BLOCK_DIM - 1 || y == N - 1)
//			{
//				for (int j = 1; j <= R; ++j)
//					for (int i = -1; i >= -R; --i)
//					{
//						int offset_x = max(0, x + i);
//						int offset_y = min(y + j, height - 1);
//						smem[y_s + j][x_s + i] = inputImage[offset_y*(N)+offset_x];
//					}
//			}
//			for (int i = -1; i >= -R; --i)
//			{
//				int offset_x = max(0, x + i);
//				smem[y_s][x_s + i] = inputImage[y*(N)+offset_x];
//			}
//		}
//		if (threadIdx.x == BLOCK_DIM - 1 || x == N - 1)
//		{
//			if (threadIdx.y == 0)
//			{
//				for (int j = -1; j >= -R; --j)
//					for (int i = 1; i <= R; ++i)
//					{
//						int offset_x = min(x + i, width - 1);
//						int offset_y = max(0, y + j);
//						smem[y_s + j][x_s + i] = inputImage[offset_y*(N)+offset_x];
//					}
//			}
//			if (threadIdx.y == BLOCK_DIM - 1 || y == N - 1)
//			{
//				for (int j = 1; j <= R; ++j)
//					for (int i = 1; i <= R; ++i)
//					{
//						int offset_x = min(x + i, width - 1);
//						int offset_y = min(y + j, height - 1);
//						smem[y_s + j][x_s + i] = inputImage[offset_y*(N)+offset_x];
//					}
//			}
//			for (int i = 1; i <= R; ++i)
//			{
//				int offset_x = min(x + i, width - 1);
//				smem[y_s][x_s + i] = inputImage[y*(N)+offset_x];
//			}
//		}
//		if (threadIdx.y == 0)
//		{
//			for (int i = -1; i >= -R; --i)
//			{
//				int offset_y = max(0, y + i);
//				smem[y_s + i][x_s] = inputImage[offset_y*(N)+x];
//			}
//		}
//		if (threadIdx.y == BLOCK_DIM - 1 || y == N - 1)
//		{
//			for (int i = 1; i <= R; ++i)
//			{
//				int offset_y = min(y + i, height - 1);
//				smem[y_s + i][x_s] = inputImage[offset_y*(N)+x];
//			}
//		}
//
//		__syncthreads();
//		const int dim = D;
//		const int k = dim*dim / 2;
//		int a = 0;
//		unsigned int window[dim*dim];
//
//
//		for (int h_offset = -R; h_offset <= R; h_offset++)
//			for (int w_offset = -R; w_offset <= R; w_offset++)
//				window[a++] = smem[y_s + h_offset][x_s + w_offset];
//
//		int low, high;
//		int median;
//		int middle, ll, hh;
//
//		low = 0; high = dim*dim - 1; median = (low + high) / 2;
//		for (;;)
//		{
//			if (high <= low)
//				break;
//			if (high == low + 1)
//			{
//				if (window[low] > window[high])
//				{
//					int temp = window[low];
//					window[low] = window[high];
//					window[high] = temp;
//				}
//				break;
//			}
//
//			middle = (low + high) / 2;
//			if (window[middle] > window[high])
//			{
//				int temp = window[middle];
//				window[middle] = window[high];
//				window[high] = temp;
//			}
//			if (window[low] > window[high])
//			{
//				int temp = window[low];
//				window[low] = window[high];
//				window[high] = temp;
//			}
//			if (window[middle] > window[low])
//			{
//				int temp = window[low];
//				window[low] = window[middle];
//				window[middle] = temp;
//			}
//			int temp = window[low + 1];
//			window[low + 1] = window[middle];
//			window[middle] = temp;
//
//			ll = low + 1;
//			hh = high;
//			for (;;)
//			{
//				do{
//					ll++;
//				} while (window[low] > window[ll]);
//				do
//				{
//					hh--;
//				} while (window[hh] > window[low]);
//				if (hh < ll)
//					break;
//				int temp = window[ll];
//				window[ll] = window[hh];
//				window[hh] = temp;
//			}
//
//			temp = window[low];
//			window[low] = window[hh];
//			window[hh] = temp;
//
//			if (hh <= median)
//				low = ll;
//			if (hh >= median)
//				high = hh - 1;
//		}
//		outputImage[y*width + x] = window[median];
//
//	}
//}

__global__ void median_Kernel_21(int* inputImage, int* outputImage, const int width, const int height, const int radius)
{
	const int dim = radius * 2 + 1;
	volatile __shared__ int smem[(1 + 2 * 10)][(T + 2 * 10)];
	//int *smem = &shared[0];
	//int *window = &shared[(blockDim.x+2*radius)*(1+2*radius)];
	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int x_p = x + radius;
	int x_o = threadIdx.x + radius;
	int y_o = radius;
	if (x >= width)
		return;

	if (threadIdx.x == 0)
		for (int j = 0; j < (1 + 2 * radius); ++j)
			for (int i = 0; i < (T + 2 * radius); ++i)
				smem[j][i] = -1;

	__syncthreads();

	for (int j = -radius; j <= radius; ++j)
		for (int i = -radius; i <= radius; ++i)
			if (smem[y_o + j][x_o + i] == -1)
				smem[y_o + j][x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_p + i];
	
	//find the offset of the current window
	//int a =  (threadIdx.x)*dim*dim;
	int a = 0;
	int window[21*21];
	//populate the window with the values around the current index
	for (int h_offset = -radius; h_offset <= radius; h_offset++)
		for (int w_offset = -radius; w_offset <= radius; w_offset++)
			window[a++] = smem[y_o + h_offset][ x_o + w_offset];
	/*
	* This Quickselect routine is based on the algorithm described in
	* "Numerical recipes in C", Second Edition,
	* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
	* This code by Nicolas Devillard - 1998. Public domain.
	*/
	int low, high;
	int median;
	int middle, ll, hh;
	low = 0; high = dim*dim - 1; median = (low + high) / 2;
	for (;;)
	{
		if (high <= low)
			break;
		if (high == low + 1)
		{
			if (window[low] > window[high])
			{
				int temp = window[low];
				window[low] = window[high];
				window[high] = temp;
			}
			break;
		}
		// Find median of low, middle and high items; swap into position low
		middle = (low + high) / 2;
		if (window[middle] > window[high])
		{
			int temp = window[middle];
			window[middle] = window[high];
			window[high] = temp;
		}
		if (window[low] > window[high])
		{
			int temp = window[low];
			window[low] = window[high];
			window[high] = temp;
		}
		if (window[middle] > window[low])
		{
			int temp = window[low];
			window[low] = window[middle];
			window[middle] = temp;
		}
		//Swap low item (now in position middle) into position (low+1)
		int temp = window[low + 1];
		window[low + 1] = window[middle];
		window[middle] = temp;

		//Nibble from each end towards middle, swapping items when stuck
		ll = low + 1;
		hh = high;
		for (;;)
		{
			do{
				ll++;
			} while (window[low] > window[ll]);
			do
			{
				hh--;
			} while (window[hh] > window[low]);
			if (hh < ll)
				break;
			int temp = window[ll];
			window[ll] = window[hh];
			window[hh] = temp;
		}

		//Swap middle item (in position low) back into correct position
		temp = window[low];
		window[low] = window[hh];
		window[hh] = temp;
		//Re-set active partition
		if (hh <= median)
			low = ll;
		if (hh >= median)
			high = hh - 1;
	}
	
	outputImage[y*width + x] = window[median];
}

__global__ void median_Kernel_19(int* inputImage, int* outputImage, const int width, const int height, const int radius)
{
	const int dim = radius * 2 + 1;
	__shared__ int smem[(1 + 2 * 9)][(T + 2 * 9)];
	//int *smem = &shared[0];
	//int *window = &shared[(blockDim.x+2*radius)*(1+2*radius)];
	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int x_p = x + radius;
	int x_o = threadIdx.x + radius;
	int y_o = radius;
	if (x >= width)
		return;

	if (threadIdx.x == 0)
		for (int j = 0; j < (1 + 2 * radius); ++j)
			for (int i = 0; i < (T + 2 * radius); ++i)
				smem[j][i] = 0;

	__syncthreads();

	for (int j = -radius; j <= radius; ++j)
		for (int i = -radius; i <= radius; ++i)
			if (smem[y_o + j][x_o + i] == 0)
				smem[y_o + j][x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_p + i];

	__syncthreads();

	//find the offset of the current window
	//int a =  (threadIdx.x)*dim*dim;
	int a = 0;
	int window[D*D];
	//populate the window with the values around the current index
	for (int h_offset = -radius; h_offset <= radius; h_offset++)
		for (int w_offset = -radius; w_offset <= radius; w_offset++)
			window[a++] = smem[y_o + h_offset][x_o + w_offset];

	/*
	* This Quickselect routine is based on the algorithm described in
	* "Numerical recipes in C", Second Edition,
	* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
	* This code by Nicolas Devillard - 1998. Public domain.
	*/
	int low, high;
	int median;
	int middle, ll, hh;
	low = 0; high = dim*dim - 1; median = (low + high) / 2;
	for (;;)
	{
		if (high <= low)
			break;
		if (high == low + 1)
		{
			if (window[low] > window[high])
			{
				int temp = window[low];
				window[low] = window[high];
				window[high] = temp;
			}
			break;
		}
		// Find median of low, middle and high items; swap into position low
		middle = (low + high) / 2;
		if (window[middle] > window[high])
		{
			int temp = window[middle];
			window[middle] = window[high];
			window[high] = temp;
		}
		if (window[low] > window[high])
		{
			int temp = window[low];
			window[low] = window[high];
			window[high] = temp;
		}
		if (window[middle] > window[low])
		{
			int temp = window[low];
			window[low] = window[middle];
			window[middle] = temp;
		}
		//Swap low item (now in position middle) into position (low+1)
		int temp = window[low + 1];
		window[low + 1] = window[middle];
		window[middle] = temp;

		//Nibble from each end towards middle, swapping items when stuck
		ll = low + 1;
		hh = high;
		for (;;)
		{
			do{
				ll++;
			} while (window[low] > window[ll]);
			do
			{
				hh--;
			} while (window[hh] > window[low]);
			if (hh < ll)
				break;
			int temp = window[ll];
			window[ll] = window[hh];
			window[hh] = temp;
		}

		//Swap middle item (in position low) back into correct position
		temp = window[low];
		window[low] = window[hh];
		window[hh] = temp;
		//Re-set active partition
		if (hh <= median)
			low = ll;
		if (hh >= median)
			high = hh - 1;
	}

	outputImage[y*width + x] = window[median];
}

__global__ void median_Kernel_17(int* inputImage, int* outputImage, const int width, const int height, const int radius)
{
	const int dim = radius * 2 + 1;
	__shared__ int smem[(1 + 2 * 8)][(T + 2 * 8)];
	//int *smem = &shared[0];
	//int *window = &shared[(blockDim.x+2*radius)*(1+2*radius)];
	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int x_p = x + radius;
	int x_o = threadIdx.x + radius;
	int y_o = radius;
	if (x >= width)
		return;

	if (threadIdx.x == 0)
		for (int j = 0; j < (1 + 2 * radius); ++j)
			for (int i = 0; i < (T + 2 * radius); ++i)
				smem[j][i] = 0;

	__syncthreads();

	for (int j = -radius; j <= radius; ++j)
		for (int i = -radius; i <= radius; ++i)
			if (smem[y_o + j][x_o + i] == 0)
				smem[y_o + j][x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_p + i];

	__syncthreads();

	//find the offset of the current window
	//int a =  (threadIdx.x)*dim*dim;
	int a = 0;
	int window[19*19];
	//populate the window with the values around the current index
	for (int h_offset = -radius; h_offset <= radius; h_offset++)
		for (int w_offset = -radius; w_offset <= radius; w_offset++)
			window[a++] = smem[y_o + h_offset][x_o + w_offset];

	/*
	* This Quickselect routine is based on the algorithm described in
	* "Numerical recipes in C", Second Edition,
	* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
	* This code by Nicolas Devillard - 1998. Public domain.
	*/
	int low, high;
	int median;
	int middle, ll, hh;
	low = 0; high = dim*dim - 1; median = (low + high) / 2;
	for (;;)
	{
		if (high <= low)
			break;
		if (high == low + 1)
		{
			if (window[low] > window[high])
			{
				int temp = window[low];
				window[low] = window[high];
				window[high] = temp;
			}
			break;
		}
		// Find median of low, middle and high items; swap into position low
		middle = (low + high) / 2;
		if (window[middle] > window[high])
		{
			int temp = window[middle];
			window[middle] = window[high];
			window[high] = temp;
		}
		if (window[low] > window[high])
		{
			int temp = window[low];
			window[low] = window[high];
			window[high] = temp;
		}
		if (window[middle] > window[low])
		{
			int temp = window[low];
			window[low] = window[middle];
			window[middle] = temp;
		}
		//Swap low item (now in position middle) into position (low+1)
		int temp = window[low + 1];
		window[low + 1] = window[middle];
		window[middle] = temp;

		//Nibble from each end towards middle, swapping items when stuck
		ll = low + 1;
		hh = high;
		for (;;)
		{
			do{
				ll++;
			} while (window[low] > window[ll]);
			do
			{
				hh--;
			} while (window[hh] > window[low]);
			if (hh < ll)
				break;
			int temp = window[ll];
			window[ll] = window[hh];
			window[hh] = temp;
		}

		//Swap middle item (in position low) back into correct position
		temp = window[low];
		window[low] = window[hh];
		window[hh] = temp;
		//Re-set active partition
		if (hh <= median)
			low = ll;
		if (hh >= median)
			high = hh - 1;
	}

	outputImage[y*width + x] = window[median];
}

__global__ void median_Kernel_15(int* inputImage, int* outputImage, const int width, const int height, const int radius)
{
	const int dim = radius * 2 + 1;
	__shared__ int smem[(1 + 2 * 7)][(T + 2 * 7)];
	//int *smem = &shared[0];
	//int *window = &shared[(blockDim.x+2*radius)*(1+2*radius)];
	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int x_p = x + radius;
	int x_o = threadIdx.x + radius;
	int y_o = radius;
	if (x >= width)
		return;

	if (threadIdx.x == 0)
		for (int j = 0; j < (1 + 2 * radius); ++j)
			for (int i = 0; i < (T + 2 * radius); ++i)
				smem[j][i] = 0;

	__syncthreads();

	for (int j = -radius; j <= radius; ++j)
		for (int i = -radius; i <= radius; ++i)
			if (smem[y_o + j][x_o + i] == 0)
				smem[y_o + j][x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_p + i];

	__syncthreads();

	//find the offset of the current window
	//int a =  (threadIdx.x)*dim*dim;
	int a = 0;
	int window[15*15];
	//populate the window with the values around the current index
	for (int h_offset = -radius; h_offset <= radius; h_offset++)
		for (int w_offset = -radius; w_offset <= radius; w_offset++)
			window[a++] = smem[y_o + h_offset][x_o + w_offset];

	/*
	* This Quickselect routine is based on the algorithm described in
	* "Numerical recipes in C", Second Edition,
	* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
	* This code by Nicolas Devillard - 1998. Public domain.
	*/
	int low, high;
	int median;
	int middle, ll, hh;
	low = 0; high = dim*dim - 1; median = (low + high) / 2;
	for (;;)
	{
		if (high <= low)
			break;
		if (high == low + 1)
		{
			if (window[low] > window[high])
			{
				int temp = window[low];
				window[low] = window[high];
				window[high] = temp;
			}
			break;
		}
		// Find median of low, middle and high items; swap into position low
		middle = (low + high) / 2;
		if (window[middle] > window[high])
		{
			int temp = window[middle];
			window[middle] = window[high];
			window[high] = temp;
		}
		if (window[low] > window[high])
		{
			int temp = window[low];
			window[low] = window[high];
			window[high] = temp;
		}
		if (window[middle] > window[low])
		{
			int temp = window[low];
			window[low] = window[middle];
			window[middle] = temp;
		}
		//Swap low item (now in position middle) into position (low+1)
		int temp = window[low + 1];
		window[low + 1] = window[middle];
		window[middle] = temp;

		//Nibble from each end towards middle, swapping items when stuck
		ll = low + 1;
		hh = high;
		for (;;)
		{
			do{
				ll++;
			} while (window[low] > window[ll]);
			do
			{
				hh--;
			} while (window[hh] > window[low]);
			if (hh < ll)
				break;
			int temp = window[ll];
			window[ll] = window[hh];
			window[hh] = temp;
		}

		//Swap middle item (in position low) back into correct position
		temp = window[low];
		window[low] = window[hh];
		window[hh] = temp;
		//Re-set active partition
		if (hh <= median)
			low = ll;
		if (hh >= median)
			high = hh - 1;
	}

	outputImage[y*width + x] = window[median];
}

__global__ void median_Kernel_13(int* inputImage, int* outputImage, const int width, const int height, const int radius)
{
	const int dim = radius * 2 + 1;
	__shared__ int smem[(1 + 2 * 6)][(T + 2 * 6)];
	//int *smem = &shared[0];
	//int *window = &shared[(blockDim.x+2*radius)*(1+2*radius)];
	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int x_p = x + radius;
	int x_o = threadIdx.x + radius;
	int y_o = radius;
	if (x >= width)
		return;

	if (threadIdx.x == 0)
		for (int j = 0; j < (1 + 2 * radius); ++j)
			for (int i = 0; i < (T + 2 * radius); ++i)
				smem[j][i] = 0;

	__syncthreads();

	for (int j = -radius; j <= radius; ++j)
		for (int i = -radius; i <= radius; ++i)
			if (smem[y_o + j][x_o + i] == 0)
				smem[y_o + j][x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_p + i];

	__syncthreads();

	//find the offset of the current window
	//int a =  (threadIdx.x)*dim*dim;
	int a = 0;
	int window[13*13];
	//populate the window with the values around the current index
	for (int h_offset = -radius; h_offset <= radius; h_offset++)
		for (int w_offset = -radius; w_offset <= radius; w_offset++)
			window[a++] = smem[y_o + h_offset][x_o + w_offset];

	/*
	* This Quickselect routine is based on the algorithm described in
	* "Numerical recipes in C", Second Edition,
	* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
	* This code by Nicolas Devillard - 1998. Public domain.
	*/
	int low, high;
	int median;
	int middle, ll, hh;
	low = 0; high = dim*dim - 1; median = (low + high) / 2;
	for (;;)
	{
		if (high <= low)
			break;
		if (high == low + 1)
		{
			if (window[low] > window[high])
			{
				int temp = window[low];
				window[low] = window[high];
				window[high] = temp;
			}
			break;
		}
		// Find median of low, middle and high items; swap into position low
		middle = (low + high) / 2;
		if (window[middle] > window[high])
		{
			int temp = window[middle];
			window[middle] = window[high];
			window[high] = temp;
		}
		if (window[low] > window[high])
		{
			int temp = window[low];
			window[low] = window[high];
			window[high] = temp;
		}
		if (window[middle] > window[low])
		{
			int temp = window[low];
			window[low] = window[middle];
			window[middle] = temp;
		}
		//Swap low item (now in position middle) into position (low+1)
		int temp = window[low + 1];
		window[low + 1] = window[middle];
		window[middle] = temp;

		//Nibble from each end towards middle, swapping items when stuck
		ll = low + 1;
		hh = high;
		for (;;)
		{
			do{
				ll++;
			} while (window[low] > window[ll]);
			do
			{
				hh--;
			} while (window[hh] > window[low]);
			if (hh < ll)
				break;
			int temp = window[ll];
			window[ll] = window[hh];
			window[hh] = temp;
		}

		//Swap middle item (in position low) back into correct position
		temp = window[low];
		window[low] = window[hh];
		window[hh] = temp;
		//Re-set active partition
		if (hh <= median)
			low = ll;
		if (hh >= median)
			high = hh - 1;
	}

	outputImage[y*width + x] = window[median];
}

__global__ void median_Kernel_11(int* inputImage, int* outputImage, const int width, const int height, const int radius)
{
	const int dim = radius * 2 + 1;
	__shared__ int smem[(1 + 2 * 5)][(T + 2 * 5)];
	//int *smem = &shared[0];
	//int *window = &shared[(blockDim.x+2*radius)*(1+2*radius)];
	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int x_p = x + radius;
	int x_o = threadIdx.x + radius;
	int y_o = radius;
	if (x >= width)
		return;

	if (threadIdx.x == 0)
		for (int j = 0; j < (1 + 2 * radius); ++j)
			for (int i = 0; i < (T + 2 * radius); ++i)
				smem[j][i] = 0;

	__syncthreads();

	for (int j = -radius; j <= radius; ++j)
		for (int i = -radius; i <= radius; ++i)
			if (smem[y_o + j][x_o + i] == 0)
				smem[y_o + j][x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_p + i];

	__syncthreads();

	//find the offset of the current window
	//int a =  (threadIdx.x)*dim*dim;
	int a = 0;
	int window[11*11];
	//populate the window with the values around the current index
	for (int h_offset = -radius; h_offset <= radius; h_offset++)
		for (int w_offset = -radius; w_offset <= radius; w_offset++)
			window[a++] = smem[y_o + h_offset][x_o + w_offset];

	/*
	* This Quickselect routine is based on the algorithm described in
	* "Numerical recipes in C", Second Edition,
	* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
	* This code by Nicolas Devillard - 1998. Public domain.
	*/
	int low, high;
	int median;
	int middle, ll, hh;
	low = 0; high = dim*dim - 1; median = (low + high) / 2;
	for (;;)
	{
		if (high <= low)
			break;
		if (high == low + 1)
		{
			if (window[low] > window[high])
			{
				int temp = window[low];
				window[low] = window[high];
				window[high] = temp;
			}
			break;
		}
		// Find median of low, middle and high items; swap into position low
		middle = (low + high) / 2;
		if (window[middle] > window[high])
		{
			int temp = window[middle];
			window[middle] = window[high];
			window[high] = temp;
		}
		if (window[low] > window[high])
		{
			int temp = window[low];
			window[low] = window[high];
			window[high] = temp;
		}
		if (window[middle] > window[low])
		{
			int temp = window[low];
			window[low] = window[middle];
			window[middle] = temp;
		}
		//Swap low item (now in position middle) into position (low+1)
		int temp = window[low + 1];
		window[low + 1] = window[middle];
		window[middle] = temp;

		//Nibble from each end towards middle, swapping items when stuck
		ll = low + 1;
		hh = high;
		for (;;)
		{
			do{
				ll++;
			} while (window[low] > window[ll]);
			do
			{
				hh--;
			} while (window[hh] > window[low]);
			if (hh < ll)
				break;
			int temp = window[ll];
			window[ll] = window[hh];
			window[hh] = temp;
		}

		//Swap middle item (in position low) back into correct position
		temp = window[low];
		window[low] = window[hh];
		window[hh] = temp;
		//Re-set active partition
		if (hh <= median)
			low = ll;
		if (hh >= median)
			high = hh - 1;
	}

	outputImage[y*width + x] = window[median];
}

__global__ void median_Kernel_9(int* inputImage, int* outputImage, const int width, const int height, const int radius)
{
	const int dim = radius * 2 + 1;
	__shared__ int smem[(1 + 2 * 4)][(T + 2 * 4)];
	//int *smem = &shared[0];
	//int *window = &shared[(blockDim.x+2*radius)*(1+2*radius)];
	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int x_p = x + radius;
	int x_o = threadIdx.x + radius;
	int y_o = radius;
	if (x >= width)
		return;

	if (threadIdx.x == 0)
		for (int j = 0; j < (1 + 2 * radius); ++j)
			for (int i = 0; i < (T + 2 * radius); ++i)
				smem[j][i] = 0;

	__syncthreads();

	for (int j = -radius; j <= radius; ++j)
		for (int i = -radius; i <= radius; ++i)
			if (smem[y_o + j][x_o + i] == 0)
				smem[y_o + j][x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_p + i];

	__syncthreads();

	//find the offset of the current window
	//int a =  (threadIdx.x)*dim*dim;
	int a = 0;
	int window[9*9];
	//populate the window with the values around the current index
	for (int h_offset = -radius; h_offset <= radius; h_offset++)
		for (int w_offset = -radius; w_offset <= radius; w_offset++)
			window[a++] = smem[y_o + h_offset][x_o + w_offset];

	/*
	* This Quickselect routine is based on the algorithm described in
	* "Numerical recipes in C", Second Edition,
	* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
	* This code by Nicolas Devillard - 1998. Public domain.
	*/
	int low, high;
	int median;
	int middle, ll, hh;
	low = 0; high = dim*dim - 1; median = (low + high) / 2;
	for (;;)
	{
		if (high <= low)
			break;
		if (high == low + 1)
		{
			if (window[low] > window[high])
			{
				int temp = window[low];
				window[low] = window[high];
				window[high] = temp;
			}
			break;
		}
		// Find median of low, middle and high items; swap into position low
		middle = (low + high) / 2;
		if (window[middle] > window[high])
		{
			int temp = window[middle];
			window[middle] = window[high];
			window[high] = temp;
		}
		if (window[low] > window[high])
		{
			int temp = window[low];
			window[low] = window[high];
			window[high] = temp;
		}
		if (window[middle] > window[low])
		{
			int temp = window[low];
			window[low] = window[middle];
			window[middle] = temp;
		}
		//Swap low item (now in position middle) into position (low+1)
		int temp = window[low + 1];
		window[low + 1] = window[middle];
		window[middle] = temp;

		//Nibble from each end towards middle, swapping items when stuck
		ll = low + 1;
		hh = high;
		for (;;)
		{
			do{
				ll++;
			} while (window[low] > window[ll]);
			do
			{
				hh--;
			} while (window[hh] > window[low]);
			if (hh < ll)
				break;
			int temp = window[ll];
			window[ll] = window[hh];
			window[hh] = temp;
		}

		//Swap middle item (in position low) back into correct position
		temp = window[low];
		window[low] = window[hh];
		window[hh] = temp;
		//Re-set active partition
		if (hh <= median)
			low = ll;
		if (hh >= median)
			high = hh - 1;
	}

	outputImage[y*width + x] = window[median];
}

__global__ void median_Kernel_7(int* inputImage, int* outputImage, const int width, const int height, const int radius)
{
	const int dim = radius * 2 + 1;
	__shared__ int smem[(1 + 2 * 3)][(T + 2 * 3)];
	//int *smem = &shared[0];
	//int *window = &shared[(blockDim.x+2*radius)*(1+2*radius)];
	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int x_p = x + radius;
	int x_o = threadIdx.x + radius;
	int y_o = radius;
	if (x >= width)
		return;

	if (threadIdx.x == 0)
		for (int j = 0; j < (1 + 2 * radius); ++j)
			for (int i = 0; i < (T + 2 * radius); ++i)
				smem[j][i] = 0;

	__syncthreads();

	for (int j = -radius; j <= radius; ++j)
		for (int i = -radius; i <= radius; ++i)
			if (smem[y_o + j][x_o + i] == 0)
				smem[y_o + j][x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_p + i];

	__syncthreads();

	//find the offset of the current window
	//int a =  (threadIdx.x)*dim*dim;
	int a = 0;
	int window[7*7];
	//populate the window with the values around the current index
	for (int h_offset = -radius; h_offset <= radius; h_offset++)
		for (int w_offset = -radius; w_offset <= radius; w_offset++)
			window[a++] = smem[y_o + h_offset][x_o + w_offset];

	/*
	* This Quickselect routine is based on the algorithm described in
	* "Numerical recipes in C", Second Edition,
	* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
	* This code by Nicolas Devillard - 1998. Public domain.
	*/
	int low, high;
	int median;
	int middle, ll, hh;
	low = 0; high = dim*dim - 1; median = (low + high) / 2;
	for (;;)
	{
		if (high <= low)
			break;
		if (high == low + 1)
		{
			if (window[low] > window[high])
			{
				int temp = window[low];
				window[low] = window[high];
				window[high] = temp;
			}
			break;
		}
		// Find median of low, middle and high items; swap into position low
		middle = (low + high) / 2;
		if (window[middle] > window[high])
		{
			int temp = window[middle];
			window[middle] = window[high];
			window[high] = temp;
		}
		if (window[low] > window[high])
		{
			int temp = window[low];
			window[low] = window[high];
			window[high] = temp;
		}
		if (window[middle] > window[low])
		{
			int temp = window[low];
			window[low] = window[middle];
			window[middle] = temp;
		}
		//Swap low item (now in position middle) into position (low+1)
		int temp = window[low + 1];
		window[low + 1] = window[middle];
		window[middle] = temp;

		//Nibble from each end towards middle, swapping items when stuck
		ll = low + 1;
		hh = high;
		for (;;)
		{
			do{
				ll++;
			} while (window[low] > window[ll]);
			do
			{
				hh--;
			} while (window[hh] > window[low]);
			if (hh < ll)
				break;
			int temp = window[ll];
			window[ll] = window[hh];
			window[hh] = temp;
		}

		//Swap middle item (in position low) back into correct position
		temp = window[low];
		window[low] = window[hh];
		window[hh] = temp;
		//Re-set active partition
		if (hh <= median)
			low = ll;
		if (hh >= median)
			high = hh - 1;
	}

	outputImage[y*width + x] = window[median];
}

__global__ void median_Kernel_5(int* inputImage, int* outputImage, const int width, const int height, const int radius)
{
	const int dim = radius * 2 + 1;
	__shared__ int smem[(1 + 2 * 2)][(T + 2 * 2)];
	//int *smem = &shared[0];
	//int *window = &shared[(blockDim.x+2*radius)*(1+2*radius)];
	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int x_p = x + radius;
	int x_o = threadIdx.x + radius;
	int y_o = radius;
	if (x >= width)
		return;

	if (threadIdx.x == 0)
		for (int j = 0; j < (1 + 2 * radius); ++j)
			for (int i = 0; i < (T + 2 * radius); ++i)
				smem[j][i] = 0;

	__syncthreads();

	for (int j = -radius; j <= radius; ++j)
		for (int i = -radius; i <= radius; ++i)
			if (smem[y_o + j][x_o + i] == 0)
				smem[y_o + j][x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_p + i];

	__syncthreads();

	//find the offset of the current window
	//int a =  (threadIdx.x)*dim*dim;
	int a = 0;
	int window[5*5];
	//populate the window with the values around the current index
	for (int h_offset = -radius; h_offset <= radius; h_offset++)
		for (int w_offset = -radius; w_offset <= radius; w_offset++)
			window[a++] = smem[y_o + h_offset][x_o + w_offset];

	/*
	* This Quickselect routine is based on the algorithm described in
	* "Numerical recipes in C", Second Edition,
	* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
	* This code by Nicolas Devillard - 1998. Public domain.
	*/
	int low, high;
	int median;
	int middle, ll, hh;
	low = 0; high = dim*dim - 1; median = (low + high) / 2;
	for (;;)
	{
		if (high <= low)
			break;
		if (high == low + 1)
		{
			if (window[low] > window[high])
			{
				int temp = window[low];
				window[low] = window[high];
				window[high] = temp;
			}
			break;
		}
		// Find median of low, middle and high items; swap into position low
		middle = (low + high) / 2;
		if (window[middle] > window[high])
		{
			int temp = window[middle];
			window[middle] = window[high];
			window[high] = temp;
		}
		if (window[low] > window[high])
		{
			int temp = window[low];
			window[low] = window[high];
			window[high] = temp;
		}
		if (window[middle] > window[low])
		{
			int temp = window[low];
			window[low] = window[middle];
			window[middle] = temp;
		}
		//Swap low item (now in position middle) into position (low+1)
		int temp = window[low + 1];
		window[low + 1] = window[middle];
		window[middle] = temp;

		//Nibble from each end towards middle, swapping items when stuck
		ll = low + 1;
		hh = high;
		for (;;)
		{
			do{
				ll++;
			} while (window[low] > window[ll]);
			do
			{
				hh--;
			} while (window[hh] > window[low]);
			if (hh < ll)
				break;
			int temp = window[ll];
			window[ll] = window[hh];
			window[hh] = temp;
		}

		//Swap middle item (in position low) back into correct position
		temp = window[low];
		window[low] = window[hh];
		window[hh] = temp;
		//Re-set active partition
		if (hh <= median)
			low = ll;
		if (hh >= median)
			high = hh - 1;
	}

	outputImage[y*width + x] = window[median];
}

__global__ void median_Kernel_3(int* inputImage, int* outputImage, const int width, const int height, const int radius)
{
	const int dim = radius * 2 + 1;
	__shared__ int smem[(1 + 2 * 1)][(T + 2 * 1)];
	//int *smem = &shared[0];
	//int *window = &shared[(blockDim.x+2*radius)*(1+2*radius)];
	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int x_p = x + radius;
	int x_o = threadIdx.x + radius;
	int y_o = radius;
	if (x >= width)
		return;

	if (threadIdx.x == 0)
		for (int j = 0; j < (1 + 2 * radius); ++j)
			for (int i = 0; i < (T + 2 * radius); ++i)
				smem[j][i] = 0;

	__syncthreads();


	//smem[0][x_o - 1] = inputImage[(0)*(width + 2 * radius) + x_p - 1]; smem[0][x_o] = inputImage[(0)*(width + 2 * radius) + x_p]; smem[0][x_o + 1] = inputImage[(0)*(width + 2 * radius) + x_p + 1];
	//smem[1][x_o - 1] = inputImage[(1)*(width + 2 * radius) + x_p - 1]; smem[1][x_o] = inputImage[(1)*(width + 2 * radius) + x_p]; smem[1][x_o + 1] = inputImage[(1)*(width + 2 * radius) + x_p + 1];
	//smem[2][x_o - 1] = inputImage[(2)*(width + 2 * radius) + x_p - 1]; smem[2][x_o] = inputImage[(2)*(width + 2 * radius) + x_p]; smem[2][x_o + 1] = inputImage[(2)*(width + 2 * radius) + x_p + 1];
	for (int j = -radius; j <= radius; ++j)
		for (int i = -radius; i <= radius; ++i)
			if (smem[y_o + j][x_o + i] == 0)
				smem[y_o + j][x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_p + i];

	__syncthreads();

	//find the offset of the current window
	//int a =  (threadIdx.x)*dim*dim;
	int a = 0;
	int window[3*3];
	//populate the window with the values around the current index
//	window[0] = smem[y_o-1][x_o-1]; window[1] = smem[y_o-1][x_o]; window[2] = smem[y_o-1][x_o+1];
//	window[3] = smem[y_o][x_o - 1]; window[4] = smem[y_o][x_o]; window[5] = smem[y_o][x_o + 1];
//	window[6] = smem[y_o + 1][x_o - 1]; window[7] = smem[y_o + 1][x_o]; window[8] = smem[y_o + 1][x_o + 1];
	for (int h_offset = -radius; h_offset <= radius; h_offset++)
		for (int w_offset = -radius; w_offset <= radius; w_offset++)
			window[a++] = smem[y_o + h_offset][x_o + w_offset];

	/*
	* This Quickselect routine is based on the algorithm described in
	* "Numerical recipes in C", Second Edition,
	* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
	* This code by Nicolas Devillard - 1998. Public domain.
	*/
	int low, high;
	int median;
	int middle, ll, hh;
	low = 0; high = dim*dim - 1; median = (low + high) / 2;
	for (;;)
	{
		if (high <= low)
			break;
		if (high == low + 1)
		{
			if (window[low] > window[high])
			{
				int temp = window[low];
				window[low] = window[high];
				window[high] = temp;
			}
			break;
		}
		// Find median of low, middle and high items; swap into position low
		middle = (low + high) / 2;
		if (window[middle] > window[high])
		{
			int temp = window[middle];
			window[middle] = window[high];
			window[high] = temp;
		}
		if (window[low] > window[high])
		{
			int temp = window[low];
			window[low] = window[high];
			window[high] = temp;
		}
		if (window[middle] > window[low])
		{
			int temp = window[low];
			window[low] = window[middle];
			window[middle] = temp;
		}
		//Swap low item (now in position middle) into position (low+1)
		int temp = window[low + 1];
		window[low + 1] = window[middle];
		window[middle] = temp;

		//Nibble from each end towards middle, swapping items when stuck
		ll = low + 1;
		hh = high;
		for (;;)
		{
			do{
				ll++;
			} while (window[low] > window[ll]);
			do
			{
				hh--;
			} while (window[hh] > window[low]);
			if (hh < ll)
				break;
			int temp = window[ll];
			window[ll] = window[hh];
			window[hh] = temp;
		}

		//Swap middle item (in position low) back into correct position
		temp = window[low];
		window[low] = window[hh];
		window[hh] = temp;
		//Re-set active partition
		if (hh <= median)
			low = ll;
		if (hh >= median)
			high = hh - 1;
	}

	outputImage[y*width + x] = window[median];
}

__global__ void median_Kernel_border_prepadded(int *inputImage, int *outputImage, const int width, const int height, const int radius)//int *inputImage, int *outputImage, unsigned int width, unsigned int height)
{
	const int dim = radius * 2 + 1;
	//extern __shared__ int shared[];
	volatile __shared__ int smem[(BLOCK_DIM + 2 * R)*(BLOCK_DIM + 2 * R)];
	//int *smem = &shared[0];
	//int *window = &shared[(blockDim.x + 2 * radius)*(blockDim.y + 2 * radius)];
	int x_s = threadIdx.x + radius;
	int y_s = threadIdx.y + radius;

	int x_a = blockIdx.x*blockDim.x + threadIdx.x + radius;
	int y_a = blockIdx.y*blockDim.y + threadIdx.y + radius;

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if (x >= width || y >= height)
		return;
	//	if (smem[y_s*(BLOCK_DIM + radius * 2) + x_s] == 0)
			smem[y_s*(BLOCK_DIM + radius * 2) + x_s] = inputImage[y_a*(width + 2 * radius) + x_a];

		//pad the left column
		if (threadIdx.x == 0)
		{
			//if it is the first row pad the top left corner
			if (threadIdx.y == 0)
			{
				for (int j = 0; j < radius; ++j)
					for (int i = 0; i < radius; ++i)
					//	if (smem[(threadIdx.y + j)*(BLOCK_DIM + radius * 2) + threadIdx.x + i] == 0)
							smem[(threadIdx.y + j)*(BLOCK_DIM + radius * 2) + threadIdx.x + i] = inputImage[(y + j)*(width + 2 * radius) + x + i];
			}
			//if it the last row pad the bottow left corner
			if (threadIdx.y == BLOCK_DIM - 1 || y == width - 1)
			{
				for (int j = 0; j < radius; ++j)
					for (int i = 0; i < radius; ++i)
					//	if (smem[(y_s + 1 + j)*(BLOCK_DIM + radius * 2) + threadIdx.x + i] == 0)
							smem[(y_s + 1 + j)*(BLOCK_DIM + radius * 2) + threadIdx.x + i] = inputImage[(y_a + 1 + j)*(width + 2 * radius) + x + i];
			}
			//pad to the left of the column
			for (int i = 0; i < radius; ++i)
			{
				//if (smem[y_s*(BLOCK_DIM + radius * 2) + threadIdx.x + i] == 0)
					smem[y_s*(BLOCK_DIM + radius * 2) + threadIdx.x + i] = inputImage[y_a*(width + 2 * radius) + x + i];
			}
		}
		//pad the right column
		if (threadIdx.x == BLOCK_DIM - 1 || x == width - 1)
		{
			//if it is the first row pad the top right corner
			if (threadIdx.y == 0)
			{
				for (int j = 0; j < radius; ++j)
					for (int i = 0; i < radius; ++i)
						//if (smem[(threadIdx.y + j)*(BLOCK_DIM + radius * 2) + x_s + 1 + i] == 0)
							smem[(threadIdx.y + j)*(BLOCK_DIM + radius * 2) + x_s + 1 + i] = inputImage[(y + j)*(width + 2 * radius) + x_a + 1 + i];
			}
			//if it is the last row pad the bottom right corner
			if (threadIdx.y == BLOCK_DIM - 1 || y == width - 1)
			{
				for (int j = 0; j < radius; ++j)
					for (int i = 0; i < radius; ++i)
						//if (smem[(y_s + 1 + j)*(BLOCK_DIM + radius * 2) + x_s + 1 + i] == 0)
							smem[(y_s + 1 + j)*(BLOCK_DIM + radius * 2) + x_s + 1 + i] = inputImage[(y_a + 1 + j)*(width + 2 * radius) + x_a + 1 + i];
			}
			//otherwise pad to the right
			for (int i = 0; i < radius; ++i)
			{
				//if (smem[y_s*(BLOCK_DIM + radius * 2) + x_s + 1 + i] == 0)
					smem[y_s*(BLOCK_DIM + radius * 2) + x_s + 1 + i] = inputImage[y_a*(width + 2 * radius) + x_a + 1 + i];
			}
		}
		//pad the top row
		if (threadIdx.y == 0)
		{
			//pad above the current index
			for (int i = 0; i < radius; ++i)
			{
			//	if (smem[(threadIdx.y + i)*(BLOCK_DIM + radius * 2) + x_s] == 0)
					smem[(threadIdx.y + i)*(BLOCK_DIM + radius * 2) + x_s] = inputImage[(y + i)*(width + 2 * radius) + x_a];
			}
		}
		//pad the bottom row
		if (threadIdx.y == BLOCK_DIM - 1 || y == width - 1)
		{
			for (int i = 0; i < radius; ++i)
			{
				//pad the below the index
				//if (smem[(y_s + 1 + i)*(BLOCK_DIM + radius * 2) + x_s] == 0)
					smem[(y_s + 1 + i)*(BLOCK_DIM + radius * 2) + x_s] = inputImage[(y_a + 1 + i)*(width + 2 * radius) + x_a];
			}
		}

		__syncthreads();
		//find the offset of the current window
		//int a = (threadIdx.x + threadIdx.y*BLOCK_DIM)*dim*dim;
		int window[D*D];
		int a = 0;
		//populate the window with the values around the current index
		for (int h_offset = -radius; h_offset <= radius; h_offset++)
			for (int w_offset = -radius; w_offset <= radius; w_offset++)
				window[a++] = smem[(y_s + h_offset) * (BLOCK_DIM + 2 * radius) + x_s + w_offset];

		int low, high;
		int median;
		int middle, ll, hh;
		/*
		* This Quickselect routine is based on the algorithm described in
		* "Numerical recipes in C", Second Edition,
		* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
		* This code by Nicolas Devillard - 1998. Public domain.
		*/
		//low = (threadIdx.x + threadIdx.y*BLOCK_DIM)*dim*dim; high = (threadIdx.x + threadIdx.y*BLOCK_DIM)*dim*dim+(dim*dim - 1); median = (low + high) / 2;
		low = 0; high = D*D - 1; median = (low + high) / 2;
		//for (;;)
		//{
		//	if (high <= low)
		//		break;
		//	if (high == low + 1)
		//	{
		//		if (window[low] > window[high])
		//		{
		//			int temp = window[low];
		//			window[low] = window[high];
		//			window[high] = temp;
		//		}
		//		break;
		//	}
		//	// Find median of low, middle and high items; swap into position low
		//	middle = (low + high) / 2;
		//	if (window[middle] > window[high])
		//	{
		//		int temp = window[middle];
		//		window[middle] = window[high];
		//		window[high] = temp;
		//	}
		//	if (window[low] > window[high])
		//	{
		//		int temp = window[low];
		//		window[low] = window[high];
		//		window[high] = temp;
		//	}
		//	if (window[middle] > window[low])
		//	{
		//		int temp = window[low];
		//		window[low] = window[middle];
		//		window[middle] = temp;
		//	}
		//	//Swap low item (now in position middle) into position (low+1)
		//	int temp = window[low+1];
		//	window[low+1] = window[middle];
		//	window[middle] = temp;

		//	//Nibble from each end towards middle, swapping items when stuck
		//	ll = low + 1;
		//	hh = high;
		//	for (;;)
		//	{
		//		do{
		//			ll++;
		//		} while (window[low] > window[ll]);
		//		do
		//		{
		//			hh--;
		//		}
		//		while (window[hh] > window[low]);
		//		if (hh < ll)
		//			break;
		//		int temp = window[ll];
		//		window[ll] = window[hh];
		//		window[hh] = temp;
		//	}

		//	//Swap middle item (in position low) back into correct position
		//	temp = window[low];
		//	window[low] = window[hh];
		//	window[hh] = temp;
		//	//Re-set active partition
		//	if (hh <= median)
		//		low = ll;
		//	if (hh >= median)
		//		high = hh - 1;
		//}

		outputImage[y*width + x] = window[median];
}

int main(int argc, char* argv[])
{
	char* name;
	int bin_dim;
	int dim;
	if (argc < 3) {
		printf("The program requires you to specify the file, bin dimension and filter radius \n\n");
		return 1;
	}
	else
	{
		name = argv[1];
		try{
			bin_dim = std::stoi(argv[2]);
		}
		catch (std::exception const & e)
		{
			printf("--- ERROR : '%s' is not a valid integer\n", argv[1]);
			return 1;
		}
		try{
			dim = std::stoi(argv[3]);
		}
		catch (std::exception const & e)
		{
			printf("--- ERROR :'%s' is not a valid integer\n", argv[2]);
			return 1;
		}
	}

	int radius = (dim - 1) / 2;
	double t1, t2;
	Grid grid_p = Grid(bin_dim, bin_dim);
	Grid cuda = grid_p;

	printf("\n================== CONFIGURATION ==================\n\n");
	printf("--- Bin dimensions \t\t: %d x %d\n", bin_dim, bin_dim);
	printf("--- Filter dimensions \t\t: %d x %d\n", dim, dim);
	printf("--- OpenMP threads \t\t: %d\n", omp_get_max_threads());

	printf("\n=================== BINNING ===================\n\n");
	printf("--- Serial Binning ---\n\n");
	int success = ReadWrite::LoadData_s(grid_p, name);
	printf("--- OMP Binning ---\n\n");
	success = ReadWrite::LoadData_omp(grid_p, name);
	if (success == -1)
		return 2;
	Grid input = grid_p;
	printf("\n================= Serial Filtering =================\n\n");
	
	t1 = omp_get_wtime();
	Grid padded = grid_p.Pad(radius);
	t2 = omp_get_wtime();
	printf("--- Time for padding \t\t: %-12.5f sec.\n", t2 - t1);

	//three slower sorting algorithms
	/*t1 = omp_get_wtime();
	Filter::m_Filter_half(padded, grid_p, dim);
	t2 = omp_get_wtime();
	printf("--- Time for half bubble sort \t: %-12.5f sec.\n", t2 - t1);
	printf("--- Count after filter \t\t: %d\n", grid_p.Count());

	t1 = omp_get_wtime();
	Filter::m_Filter_quickselect(padded, grid_p, dim);
	t2 = omp_get_wtime();
	printf("--- Time for Quick Select \t: %-12.5f sec.\n", t2 - t1);
	printf("--- Count after filter \t\t: %d\n", grid_p.Count());*/
	
	t1 = omp_get_wtime();
	//Filter::m_Filter_quickselect2(padded, grid_p, dim);
	t2 = omp_get_wtime();
	printf("--- Time for Quick Select \t: %-12.5f sec.\n", t2 - t1);
	printf("--- Count after filter \t\t: %d\n", grid_p.Count());

	//padded.Print();
	printf("\n================== Cuda Filtering ==================\n\n");
	hipError_t cudaStatus;

	/*-------------------
	*	The two commented out kernels use constant values for the grid
	*	and filter size
	---------------------*/

	/*cudaStatus = median_filter(cuda, input, R, 1);
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "addWithCuda failed!");
	return 1;
	}
	printf("--- Count after filter \t\t: %d\n", cuda.Count());
	
	cudaStatus = median_filter(cuda, input, R, 2);
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "addWithCuda failed!");
	return 1;
	}
	printf("--- Count after filter \t\t: %d\n", cuda.Count());*/
	cudaStatus = median_filter(cuda, padded, radius, 3);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	printf("--- Count after filter \t\t: %d\n", cuda.Count());

	printf("\n===================== Writing csv =====================\n\n");
	printf("--- Writing unfiltered grid\n");
	//ReadWrite::WriteData(input, "unfiltered.csv");
	//printf("--- Writing serial grid\n");
	//ReadWrite::WriteData(grid_p, "serial.csv");
	printf("--- Writing cuda grid\n");
	//ReadWrite::WriteData(cuda, "cuda.csv");

	printf("\n======================= Grids ======================\n\n");
	if (bin_dim < 20)
	{
		printf("--- Initial Grid ---\n");
		input.Print();
		printf("---- Serial Grid ---\n");
		grid_p.Print();
		printf("----- Cuda Grid ----\n");
		cuda.Print();
	}
	else
	{
		printf("--- Initial Subset ---\n");
		input.PrintRange(0, 20, 0, 20);
		printf("---- Serial Subset ---\n");
		grid_p.PrintRange(0, 20, 0, 20);
		printf("----- Cuda Subset ----\n");
		cuda.PrintRange(0, 20, 0, 20);
	}


    //hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to apply a median filter
hipError_t median_filter(Grid &output, const Grid &input, const int filter_radius, int padding_type)
{

    int *dev_i = 0;
    int *dev_o = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for input array.
	cudaStatus = hipMalloc((void**)&dev_i, input.rows*input.cols * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	cudaStatus = hipMalloc((void**)&dev_o, output.rows*output.cols * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_i, input.grid, input.rows*input.cols * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//cudaStatus = hipBindTexture(NULL, tex, dev_i, input.rows*input.cols*sizeof(int));
	
	cudaThreadSetLimit(hipLimitMallocHeapSize, (BLOCK_DIM + 2 * filter_radius)*(BLOCK_DIM + 2 * filter_radius) * (filter_radius * 2 + 1)*(filter_radius * 2 + 1) * sizeof(int));

	dim3 grid;
	dim3 threads;
	dim3 gr, th;
	if (padding_type == 1)
	{
//		grid = dim3((N + TILE_W - 1) / TILE_W, (N + TILE_H - 1) / TILE_W);
//		threads = dim3(BLOCK_H, BLOCK_W);
	}
	else
	{
		threads = dim3(BLOCK_DIM, BLOCK_DIM);
		grid = dim3((output.cols + threads.x - 1) / threads.x, (output.rows + threads.y - 1) / threads.y);
		th = dim3(T, 1);
		gr = dim3((output.cols + th.x - 1) / th.x, output.rows);
	}
	dim3 grid_warmup(1, 1);

	if (padding_type == 1)
	{
		//		median_Kernel_threads_padding <<<grid_warmup, threads>>>(dev_i, dev_o, output.cols, output.rows);
	}
	else if (padding_type == 2)
	{
		//	median_Kernel_border_padding <<<grid_warmup, threads>>>(dev_i, dev_o, output.cols, output.rows);
	}
	else
	{ 
		//median_Kernel_border_prepadded << <grid_warmup, threads, (threads.x +2*filter_radius)*(threads.y+2*filter_radius)*sizeof(int) >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
		//median_Kernel_texture << <grid_warmup, th, ((T + 2 * filter_radius)*(1 + 2 * filter_radius) + (T*(filter_radius * 2 + 1)*(filter_radius * 2 + 1)))*sizeof(int) >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
	}

	hipEventRecord(start);

    // launch the different kernels
	if (padding_type == 1)
	{
		//median_Kernel_threads_padding << <grid_warmup, threads >> >(dev_i, dev_o, output.cols, output.rows);
	}
	else if (padding_type == 2)
	{
	//	median_Kernel_border_padding << <grid, threads >> >(dev_i, dev_o, output.cols, output.rows);
	}
	else
	{
		//median_Kernel_border_prepadded << <grid, threads/*, ((threads.x + 2 * filter_radius)*(threads.y + 2 * filter_radius) + threads.x*threads.y*(filter_radius * 2 + 1)*(filter_radius * 2 + 1))*sizeof(int)*/ >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
		if (filter_radius == 10)
		{
			median_Kernel_21 << <grid_warmup, th >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);

			median_Kernel_21 << <gr, th >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
		}
		else if (filter_radius == 9)
			median_Kernel_19 << <gr, th >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
		else if (filter_radius == 8)
			median_Kernel_17 << <gr, th >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
		else if (filter_radius == 7)
			median_Kernel_15 << <gr, th >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
		else if (filter_radius == 6)
			median_Kernel_13 << <gr, th >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
		else if (filter_radius == 5)
			median_Kernel_11 << <gr, th >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
		else if (filter_radius == 4)
			median_Kernel_9 << <gr, th >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
		else if (filter_radius == 3)
			median_Kernel_7 << <gr, th >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
		else if (filter_radius == 2)
			median_Kernel_5 << <gr, th >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
		else if (filter_radius == 1)
			median_Kernel_3 << <gr, th >> >(dev_i, dev_o, output.cols, output.rows, filter_radius);
	
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("--- Cuda Quick Select \t: %-12.5f sec\n", milliseconds/1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output.grid, dev_o, output.rows*output.cols * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	
Error:
//1	hipUnbindTexture(tex);
    hipFree(dev_i);
    hipFree(dev_o);

    return cudaStatus;
}

//the two kernels below use constant values
//__global__ void median_Kernel_border_padding(int *inputImage, int *outputImage, int width, int height)//int *inputImage, int *outputImage, unsigned int width, unsigned int height)
//{
//	__shared__ int smem[(BLOCK_DIM + 2 * R)][(BLOCK_DIM + 2 * R)];
//	int x_s = threadIdx.x + R;
//	int y_s = threadIdx.y + R;
//
//	int x = blockIdx.x*blockDim.x + threadIdx.x;
//	int y = blockIdx.y*blockDim.y + threadIdx.y;
//	if (x < N && y < N)
//	{
//		smem[y_s][x_s] = inputImage[y*(N)+x];
//
//		if (threadIdx.x == 0)
//		{
//			if (threadIdx.y == 0)
//			{
//				for (int j = -1; j >= -R; --j)
//					for (int i = -1; i >= -R; --i)
//					{
//						int offset_x = max(0, x + i);
//						int offset_y = max(0, y + j);
//						smem[y_s + j][x_s + i] = inputImage[offset_y*(N)+offset_x];
//					}
//
//			}
//			if (threadIdx.y == BLOCK_DIM - 1 || y == N - 1)
//			{
//				for (int j = 1; j <= R; ++j)
//					for (int i = -1; i >= -R; --i)
//					{
//						int offset_x = max(0, x + i);
//						int offset_y = min(y + j, height - 1);
//						smem[y_s + j][x_s + i] = inputImage[offset_y*(N)+offset_x];
//					}
//			}
//			for (int i = -1; i >= -R; --i)
//			{
//				int offset_x = max(0, x + i);
//				smem[y_s][x_s + i] = inputImage[y*(N)+offset_x];
//			}
//		}
//		if (threadIdx.x == BLOCK_DIM - 1 || x == N - 1)
//		{
//			if (threadIdx.y == 0)
//			{
//				for (int j = -1; j >= -R; --j)
//					for (int i = 1; i <= R; ++i)
//					{
//						int offset_x = min(x + i, width - 1);
//						int offset_y = max(0, y + j);
//						smem[y_s + j][x_s + i] = inputImage[offset_y*(N)+offset_x];
//					}
//			}
//			if (threadIdx.y == BLOCK_DIM - 1 || y == N - 1)
//			{
//				for (int j = 1; j <= R; ++j)
//					for (int i = 1; i <= R; ++i)
//					{
//						int offset_x = min(x + i, width - 1);
//						int offset_y = min(y + j, height - 1);
//						smem[y_s + j][x_s + i] = inputImage[offset_y*(N)+offset_x];
//					}
//			}
//			for (int i = 1; i <= R; ++i)
//			{
//				int offset_x = min(x + i, width - 1);
//				smem[y_s][x_s + i] = inputImage[y*(N)+offset_x];
//			}
//		}
//		if (threadIdx.y == 0)
//		{
//			for (int i = -1; i >= -R; --i)
//			{
//				int offset_y = max(0, y + i);
//				smem[y_s + i][x_s] = inputImage[offset_y*(N)+x];
//			}
//		}
//		if (threadIdx.y == BLOCK_DIM - 1 || y == N - 1)
//		{
//			for (int i = 1; i <= R; ++i)
//			{
//				int offset_y = min(y + i, height - 1);
//				smem[y_s + i][x_s] = inputImage[offset_y*(N)+x];
//			}
//		}
//
//		__syncthreads();
//		const int dim = D;
//		const int k = dim*dim / 2;
//		int a = 0;
//		unsigned int window[dim*dim];
//
//
//		for (int h_offset = -R; h_offset <= R; h_offset++)
//			for (int w_offset = -R; w_offset <= R; w_offset++)
//				window[a++] = smem[y_s + h_offset][x_s + w_offset];
//
//		int low, high;
//		int median;
//		int middle, ll, hh;
//
//		low = 0; high = dim*dim - 1; median = (low + high) / 2;
//		for (;;)
//		{
//			if (high <= low)
//				break;
//			if (high == low + 1)
//			{
//				if (window[low] > window[high])
//				{
//					int temp = window[low];
//					window[low] = window[high];
//					window[high] = temp;
//				}
//				break;
//			}
//
//			middle = (low + high) / 2;
//			if (window[middle] > window[high])
//			{
//				int temp = window[middle];
//				window[middle] = window[high];
//				window[high] = temp;
//			}
//			if (window[low] > window[high])
//			{
//				int temp = window[low];
//				window[low] = window[high];
//				window[high] = temp;
//			}
//			if (window[middle] > window[low])
//			{
//				int temp = window[low];
//				window[low] = window[middle];
//				window[middle] = temp;
//			}
//			int temp = window[low + 1];
//			window[low + 1] = window[middle];
//			window[middle] = temp;
//
//			ll = low + 1;
//			hh = high;
//			for (;;)
//			{
//				do{
//					ll++;
//				} while (window[low] > window[ll]);
//				do
//				{
//					hh--;
//				} while (window[hh] > window[low]);
//				if (hh < ll)
//					break;
//				int temp = window[ll];
//				window[ll] = window[hh];
//				window[hh] = temp;
//			}
//
//			temp = window[low];
//			window[low] = window[hh];
//			window[hh] = temp;
//
//			if (hh <= median)
//				low = ll;
//			if (hh >= median)
//				high = hh - 1;
//		}
//		outputImage[y*width + x] = window[median];
//
//	}
//}
//
//__global__ void median_Kernel_threads_padding(int *inputImage, int *outputImage, int width, int height)//int *inputImage, int *outputImage, unsigned int width, unsigned int height)
//{
//
//	__shared__ int smem[BLOCK_W * BLOCK_H];
//
//	int x = blockIdx.x*TILE_W + threadIdx.x - R;
//
//	int y = blockIdx.y*TILE_H + threadIdx.y - R;
//
//	int a = x;
//	int b = y;
//
//	int c = blockIdx.x*blockDim.x + threadIdx.x;
//	int d = blockIdx.y*blockDim.y + threadIdx.y;
//
//	{
//		x = max(0, x);
//
//		x = min(x, width - 1);
//
//		y = max(y, 0);
//
//		y = min(y, height - 1);
//
//
//
//		unsigned int index = y*width + x;
//
//		unsigned int bindex = threadIdx.y * blockDim.y + threadIdx.x;
//
//		smem[bindex] = inputImage[index];
//
//		__syncthreads();
//		if ((threadIdx.x >= R) && (threadIdx.x < BLOCK_W - R) && (threadIdx.y >= R) && (threadIdx.y < BLOCK_W - R) && (a < N && b < N))
//
//		{
//
//			const int dim = D;
//			const int k = dim*dim / 2;
//			int a = 0;
//			unsigned int window[dim*dim];
//
//
//			for (int h_offset = -R; h_offset <= R; h_offset++)
//				for (int w_offset = -R; w_offset <= R; w_offset++)
//					window[a++] = smem[bindex + h_offset*BLOCK_W + w_offset];
//			int low, high;
//			int median;
//			int middle, ll, hh;
//
//			low = 0; high = dim*dim - 1; median = (low + high) / 2;
//			for (;;)
//			{
//				if (high <= low)
//					break;
//				if (high == low + 1)
//				{
//					if (window[low] > window[high])
//					{
//						int temp = window[low];
//						window[low] = window[high];
//						window[high] = temp;
//					}
//					break;
//				}
//
//				middle = (low + high) / 2;
//				if (window[middle] > window[high])
//				{
//					int temp = window[middle];
//					window[middle] = window[high];
//					window[high] = temp;
//				}
//				if (window[low] > window[high])
//				{
//					int temp = window[low];
//					window[low] = window[high];
//					window[high] = temp;
//				}
//				if (window[middle] > window[low])
//				{
//					int temp = window[low];
//					window[low] = window[middle];
//					window[middle] = temp;
//				}
//				int temp = window[low + 1];
//				window[low + 1] = window[middle];
//				window[middle] = temp;
//
//				ll = low + 1;
//				hh = high;
//				for (;;)
//				{
//					do{
//						ll++;
//					} while (window[low] > window[ll]);
//					do
//					{
//						hh--;
//					} while (window[hh] > window[low]);
//					if (hh < ll)
//						break;
//					int temp = window[ll];
//					window[ll] = window[hh];
//					window[hh] = temp;
//				}
//
//				temp = window[low];
//				window[low] = window[hh];
//				window[hh] = temp;
//
//				if (hh <= median)
//					low = ll;
//				if (hh >= median)
//					high = hh - 1;
//			}
//
//			outputImage[index] = window[median];
//
//		}
//
//	}
//}

//these were early attemps, windows timeout cause a lot of self doubt

//__global__ void addKernel(int *inputImage, int *outputImage, int width, int height)//int *inputImage, int *outputImage, unsigned int width, unsigned int height)
//{
//	/*int from = 0, to = dim*dim - 1;
//	while (from < to)
//	{
//	int r = from, w = to;
//	int mid = window[(r + w) / 2];
//
//	while (r < w)
//	{
//	if (window[r] >= mid)
//	{
//	int temp = window[w];
//	window[w] = window[r];
//	window[r] = temp;
//	w--;
//	}
//	else
//	{
//	r++;
//	}
//	}
//	if (window[r] > mid)
//	{
//	r--;
//	}
//	if (k <= r)
//	{
//	to = r;
//	}
//	else
//	{
//	from = r + 1;
//	}
//	}*/
//
//	//int idc, val, min, max, inf, equal, sup, mxinf, minsup, estim;
//
	//int ib = threadIdx.y;
	//int jb = threadIdx.x;
	//int idx_h = __mul24(ib+R, blockDim.x+2) + jb + 1;
	//int offset = __mul24(blockDim.x, R);

	//int j = __mul24(blockIdx.x, blockDim.x) + jb;
	//int i = __mul24(blockIdx.y, blockDim.y) + ib;
	//if (i < N && j < N)
	//{
	//	/*extern*/ __shared__ int buff[40 * 40];
	//	buff[idx_h] = inputImage[i * N + j];
	//	if (ib < R)
	//	{
	//		buff[idx_h - offset] = 0;//padding
	//	}
	//	else if (ib >= (blockDim.y-R))
	//	{
	//		buff[idx_h + offset] = 0;//padding
	//	}
	//	__syncthreads();
	//	printf("dim[%d][%d] | %-10d : %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d \n", i, j, inputImage[(i) * N + j],
	//		buff[idx_h-offset-1], buff[idx_h-offset], buff[idx_h-offset+1],
	//		buff[idx_h-1], buff[idx_h], buff[idx_h+1], 
	//		buff[idx_h+offset-1], buff[idx_h+offset], buff[idx_h+offset]);
	//	min = max = buff[ib * blockDim.x + jb];

	//	/*for (idc = 0; idc < 2 * R + 1; ++idc)
	//	{
	//	val = buff[__mul24(ib + idc, blockDim.x) + jb];
	//	if (val < min) min = val;
	//	if (val > max) max = val;
	//	}

	//	while (1)
	//	{
	//	estim = (min + max) / 2;
	//	inf = sup = equal = 0;
	//	mxinf = min;
	//	minsup = max;
	//	for (idc = 0; idc < 2 * R + 1; ++idc)
	//	{
	//	val = buff[__mul24(ib + idc, blockDim.x) + jb];
	//	if (val < estim)
	//	{
	//	inf++;
	//	if (val > mxinf) mxinf = val;
	//	}
	//	else if (val > estim)
	//	{
	//	sup++;
	//	if (val < minsup) minsup = val;
	//	}
	//	else equal++;
	//	}
	//	if ((inf <= (R + 1)) && (sup <= (R + 1))) break;
	//	else if (inf > sup) max = mxinf;
	//	else min = minsup;
	//	}
	//	if (inf >= R + 1) val = mxinf;
	//	else if (inf + equal >= R + 1) val = estim;
	//	else val = minsup;*/

	//	outputImage[__mul24(j, N) + i] = 0;//val;
	//}
//
//	//	__shared__ int smem[16 * 16];
//	//
//	//	int x = blockIdx.x*TILE_W + threadIdx.x - R;
//	//
//	//	int y = blockIdx.y*TILE_H + threadIdx.y - R;
//	//
//	//	//clamp to edge of image
//	////	if (blockIdx.x*blockDim.x + threadIdx.x < N + R && blockIdx.y*blockDim.y + threadIdx.y < N + R)
//	//	//{
//	//		x = max(0, x);
//	//
//	//		x = min(x, width - 1);
//	//
//	//		y = max(y, 0);
//	//
//	//		y = min(y, height - 1);
//	//
//	//
//	//
//	//		unsigned int index = y*width + x;
//	//
//	//		unsigned int bindex = threadIdx.y * blockDim.y + threadIdx.x;
//	//
//	//
//	//
//	//		//each thread copies its pixel of the block to shared memory
//	//
//	//		smem[bindex] = inputImage[index];
//	//
//	//		__syncthreads();
//	//
//	//		if ((threadIdx.x >= R) && (threadIdx.x < (BLOCK_W - R)) &&
//	//			(threadIdx.y >= R) && (threadIdx.y < (BLOCK_H - R)))
//	//		{
//	//			
//	//			min = max = smem[ib * blockDim.x + jb];
//	//
//	//				for (idc = 0; idc < 2 * R + 1; ++idc)
//	//				{
//	//					val = smem[__mul24(ib + idc, blockDim.x) + jb];
//	//				if (val < min) min = val;
//	//				if (val > max) max = val;
//	//				}
//	//
//	//				while (1)
//	//				{
//	//				estim = (min + max) / 2;
//	//				inf = sup = equal = 0;
//	//				mxinf = min;
//	//				minsup = max;
//	//				for (idc = 0; idc < 2 * R + 1; ++idc)
//	//				{
//	//					val = smem[__mul24(ib + idc, blockDim.x) + jb];
//	//				if (val < estim)
//	//				{
//	//				inf++;
//	//				if (val > mxinf) mxinf = val;
//	//				}
//	//				else if (val > estim)
//	//				{
//	//				sup++;
//	//				if (val < minsup) minsup = val;
//	//				}
//	//				else equal++;
//	//				}
//	//				if ((inf <= (R + 1)) && (sup <= (R + 1))) break;
//	//				else if (inf > sup) max = mxinf;
//	//				else min = minsup;
//	//				}
//	//				if (inf >= R + 1) val = mxinf;
//	//				else if (inf + equal >= R + 1) val = estim;
//	//				else val = minsup;
//	//
//	//			/*int      val, i, less, greater, equal,  min, max, guess, maxltguess, mingtguess;
//	//
//	//			min = max = smem[bindex];
//	//			for (i =-1; i<=1; i++) {
//	//			val = smem[__mul24(threadIdx.y + i, blockDim.x) + threadIdx.x];
//	//			if (val<min) min = val;
//	//			if (val> max) max = val;
//	//			}
//	//
//	//			while (1) {
//	//				guess = (min + max) / 2;
//	//				less = 0; greater = 0; equal = 0;
//	//				maxltguess = min;
//	//				mingtguess = max;
//	//				for (i = -1; i<=1; i++) {
//	//					val = smem[__mul24(threadIdx.y + i, blockDim.x) + threadIdx.x];
//	//					if (val<guess) {
//	//						less++;
//	//						if (val>maxltguess) maxltguess = val;
//	//					}
//	//					else if (val>guess) {
//	//						greater++;
//	//					if (val<mingtguess) mingtguess = val;
//	//					}
//	//					else equal++;
//	//				}
//	//				if (less <= (R + 1) && greater <= (R + 1)) break;
//	//				else if (less>greater) max = maxltguess;
//	//				else min = mingtguess;
//	//			}
//	//			if (less >= (R + 1)) val = maxltguess;
//	//			else if (less + equal >= (R + 1)) val = guess;
//	//			else val = mingtguess;*/
//	//			/*printf("dim[%d][%d] | %-10d : %-10d, %-10d, %-10d, %-10d, %-10d, %-10d, %-10d, %-10d, %-10d \n", y, x, inputImage[index],
//	//				smem[bindex - blockDim.x - 1], smem[bindex - blockDim.x], smem[bindex - blockDim.x + 1],
//	//				smem[bindex - 1], smem[bindex], smem[bindex + 1],
//	//				smem[bindex + blockDim.x - 1], smem[bindex + blockDim.x], smem[bindex + blockDim.x + 1]);*/
//	//			//outputImage[index] =  smem[bindex];
//	//			outputImage[index] = val;
//	//
//	//		}
//	//}
//
//	//extern __shared__ int data[];
//	//__shared__ int data[(R * 2 + 1) * (T + 2 * R)];
//	//int col = blockIdx.x * blockDim.x + threadIdx.x;
//	//int row = blockIdx.y * blockDim.y + threadIdx.y;
//	//int d_col = col + R;
//	//int d_row = blockIdx.y;
//
//	//if (row < N && col < N)
//	//{
//
//	//	for (int r = -R; r <= R; ++r)
//	//	{
//	//		//col += r;
//	//		int i_row = row + r;
//	//		//col = max(0, col);
//
//	//		//col = min(col, N - 1);
//
//	//		i_row = max(i_row, 0);
//
//	//		i_row = min(i_row, N - 1);
//	//		//data[1028 + col] = 0; //inputImage[row*N + col];
//	//		data[(R + r) * (T + 2 * R) + d_col] = inputImage[i_row*N + col];
//	//		if (col == 0)
//	//		{
//	//			for (int i = 0; i <= R;++i)
//	//				data[(R + r) * (T + 2 * R) + col+i] = inputImage[i_row*N + col];
//	//		}
//	//		if (col == N - 1 || col == blockDim.x)
//	//		{
//	//			for (int i = 0; i <= R; ++i)
//	//				data[(R + r) * (T + 2 * R) + d_col + i] = inputImage[i_row*N + col];
//	//		}
//	//	}
//	//	__syncthreads();
//	/*printf("dim : %d | [%d][%d] | %-10d : %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d \n", blockDim.y, d_row, col, inputImage[(row) * N + col],
//	data[(0) * 514 + d_col - 1], data[(0) * 514 + d_col], data[(0) * 514 + d_col + 1],
//	data[(1) * 514 + d_col - 1], data[(1) * 514 + d_col], data[(1) * 514 + d_col + 1],
//	data[(2) * 514 + d_col - 1], data[(2) * 514 + d_col], data[(2) * 514 + d_col + 1]);*/
//	//printf("dim : %d | [%d][%d] | %-10d : %-9d, %-9d, %-9d\n", blockDim.y, d_row, N, inputImage[(row) * N + col], data[(0) * 514 + N], data[(1)* 514 + N], data[(2) * 514 + N]);
//	//int k = 0;
//	//int *window = new int[D*D];
//	//for (int dy = -R; dy <= R; ++dy)
//	//{
//	//	for (int dx = -R; dx <= R; ++dx)
//	//	{
//	//		window[k++] = data[(R + dy) * (T + 2 * R) + d_col + dx];
//	//	}
//	//}
//	//for (int j = 0; j < (D*D+1)/2; ++j)
//	//{
//	//	//   find position of minimum element
//	//	int min = j;
//	//	for (int l = j + 1; l < (D*D); ++l)
//	//		if (window[l] < window[min])
//	//			min = l;
//	//	const int temp = window[j];
//	//	window[j] = window[min];
//	//	window[min] = temp;
//	//}
//	/*int      val, i, less, greater, equal,  min, max, guess, maxltguess, mingtguess;
//
//	min = max = data[__mul24(threadIdx.y + R, blockDim.x + 2 * R) + threadIdx.x + R];
//	for (i =0; i<D; i++) {
//	val = data[__mul24(threadIdx.y + i, blockDim.x) + threadIdx.x];
//	if (val<min) min = val;
//	if (val> max) max = val;
//	}
//	while (1) {
//	guess = (min + max) / 2;
//	less = 0; greater = 0; equal = 0;
//	maxltguess = min;
//	mingtguess = max;
//	for (i = 0; i<D; i++) {
//	val = data[__mul24(threadIdx.y + i, blockDim.x) + threadIdx.x];
//	if (val<guess) {
//	less++;
//	if (val>maxltguess) maxltguess = val;
//	}
//	else if (val>guess) {
//	greater++;
//	if (val<mingtguess) mingtguess = val;
//	}
//	else equal++;
//	}
//	if (less <= (R + 1) && greater <= (R + 1)) break;
//	else if (less>greater) max = maxltguess;
//	else min = mingtguess;
//	}
//	if (less >= (R + 1)) val = maxltguess;
//	else if (less + equal >= (R + 1)) val = guess;
//	else val = mingtguess;*/
//
//	//outputImage[(row)* N + col] = val;
//	//outputImage[(row)* N + col] = data[(R)* (T + 2 * R) + d_col/*(1) * 514 + d_col*/];
//	//outputImage[(row)* N + col] = 1;// window[(D*D) / 2];
//	//}
//}

//__global__ void median_Kernel_1d(int* inputImage, int* outputImage, const int width, const int height, const int radius)
//{
//	const int dim = radius * 2 + 1;
//	extern __shared__ int shared[];
//	int *smem = &shared[0];
//	int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
//	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
//	int x = blockIdx.x*blockDim.x + threadIdx.x;
//	int y = blockIdx.y;
//	int x_o = threadIdx.x + radius;
//	int y_o = radius;
//	if (x >= width)
//		return;
//	//for (int j = -radius; j <= radius; ++j)
//	//for (int i = -radius; i <= radius; ++i)
//	//if (smem[(y_o + j)*(blockDim.x + 2 * radius) + x_o + i] == 0)
//	//smem[(y_o + j)*(blockDim.x + 2 * radius) + x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_o + i];
//	for (int j = -radius; j <= radius; ++j)
//		if (smem[(y_o + j)*(blockDim.x + 2 * radius) + x_o] == 0)
//			smem[(y_o + j)*(blockDim.x + 2 * radius) + x_o] = inputImage[(y + radius + j)*(width + 2 * radius) + x_o];
//	if (threadIdx.x == 0)
//		for (int j = -radius; j <= radius; ++j)
//			for (int i = -radius; i < 0; ++i)
//				if (smem[(y_o + j)*(blockDim.x + 2 * radius) + x_o + i] == 0)
//					smem[(y_o + j)*(blockDim.x + 2 * radius) + x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_o + i];
//	if (threadIdx.x == blockDim.x - 1 || threadIdx.x == width - 1)
//		for (int j = -radius; j <= radius; ++j)
//			for (int i = 1; i <= radius; ++i)
//				if (smem[(y_o + j)*(blockDim.x + 2 * radius) + x_o + i] == 0)
//					smem[(y_o + j)*(blockDim.x + 2 * radius) + x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_o + i];
//
//	__syncthreads();
//	//find the offset of the current window
//	int a = (threadIdx.x)*dim*dim;
//	//populate the window with the values around the current index
//	for (int h_offset = -radius; h_offset <= radius; h_offset++)
//		for (int w_offset = -radius; w_offset <= radius; w_offset++)
//			window[a++] = smem[(y_o + h_offset) * (blockDim.x + 2 * radius) + x_o + w_offset];
//
//	int low, high;
//	int median;
//	int middle, ll, hh;
//	/*
//	* This Quickselect routine is based on the algorithm described in
//	* "Numerical recipes in C", Second Edition,
//	* Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
//	* This code by Nicolas Devillard - 1998. Public domain.
//	*/
//	low = (threadIdx.x)*dim*dim; high = (threadIdx.x)*dim*dim + (dim*dim - 1); median = (low + high) / 2;
//	for (;;)
//	{
//		if (high <= low)
//			break;
//		if (high == low + 1)
//		{
//			if (window[low] > window[high])
//			{
//				int temp = window[low];
//				window[low] = window[high];
//				window[high] = temp;
//			}
//			break;
//		}
//		// Find median of low, middle and high items; swap into position low
//		middle = (low + high) / 2;
//		if (window[middle] > window[high])
//		{
//			int temp = window[middle];
//			window[middle] = window[high];
//			window[high] = temp;
//		}
//		if (window[low] > window[high])
//		{
//			int temp = window[low];
//			window[low] = window[high];
//			window[high] = temp;
//		}
//		if (window[middle] > window[low])
//		{
//			int temp = window[low];
//			window[low] = window[middle];
//			window[middle] = temp;
//		}
//		//Swap low item (now in position middle) into position (low+1)
//		int temp = window[low + 1];
//		window[low + 1] = window[middle];
//		window[middle] = temp;
//
//		//Nibble from each end towards middle, swapping items when stuck
//		ll = low + 1;
//		hh = high;
//		for (;;)
//		{
//			do{
//				ll++;
//			} while (window[low] > window[ll]);
//			do
//			{
//				hh--;
//			} while (window[hh] > window[low]);
//			if (hh < ll)
//				break;
//			int temp = window[ll];
//			window[ll] = window[hh];
//			window[hh] = temp;
//		}
//
//		//Swap middle item (in position low) back into correct position
//		temp = window[low];
//		window[low] = window[hh];
//		window[hh] = temp;
//		//Re-set active partition
//		if (hh <= median)
//			low = ll;
//		if (hh >= median)
//			high = hh - 1;
//	}
//
//	outputImage[y*width + x] = window[median];
//}


//__global__ void median_Kernel_torben(int* inputImage, int* outputImage, const int width, const int height, const int radius)
//{
//	const int dim = radius * 2 + 1;
//	__shared__ int smem[(1 + 2 * R)][(T + 2 * R)];
//	//int *smem = &shared[0];
//	//int *window = &shared[(blockDim.x+2*radius)*(1+2*radius)];
//	//int *window = &shared[(blockDim.x + 2 * radius)*(1 + 2 * radius)];
//	int x = blockIdx.x*blockDim.x + threadIdx.x;
//	int y = blockIdx.y;
//	int x_p = x + radius;
//	int x_o = threadIdx.x + radius;
//	int y_o = radius;
//	if (x >= width)
//		return;
//
//	if (threadIdx.x == 0)
//		for (int j = 0; j < (1 + 2 * R); ++j)
//			for (int i = 0; i < (T + 2 * R); ++i)
//				smem[j][i] = 0;
//
//	__syncthreads();
//
//	for (int j = -radius; j <= radius; ++j)
//		for (int i = -radius; i <= radius; ++i)
//			if (smem[y_o + j][x_o + i] == 0)
//				smem[y_o + j][x_o + i] = inputImage[(y + radius + j)*(width + 2 * radius) + x_p + i];
//
//	__syncthreads();
//	/*
//
//	* The following code is public domain.
//
//	* Algorithm by Torben Mogensen, implementation by N. Devillard.
//
//	* This code in public domain.
//
//	*/
//
//
//	int i, less, greater, equal;
//	int min, max, guess, maxltguess, mingtguess;
//	min = max = smem[0][threadIdx.x];
//	for (i = 1; i < dim*dim; i++) {
//		int row = i / dim;
//		int col = i%dim;
//		if (smem[row][threadIdx.x + col] < min) min = smem[row][threadIdx.x + col];
//		if (smem[row][threadIdx.x + col] > max) max = smem[row][threadIdx.x + col];
//	}
//	while (1) {
//		guess = (min + max) / 2;
//		less = 0; greater = 0; equal = 0;
//		maxltguess = min;
//		mingtguess = max;
//		for (i = 0; i<dim*dim; i++)
//		{
//			int row = i / dim;
//			int col = i%dim;
//			if (smem[row][threadIdx.x + col]<guess)
//			{
//				less++;
//				if (smem[row][threadIdx.x + col]>maxltguess) maxltguess = smem[row][threadIdx.x + col];
//			}
//			else if (smem[row][threadIdx.x + col]>guess)
//			{
//				greater++;
//
//				if (smem[row][threadIdx.x + col] <mingtguess) mingtguess = smem[row][threadIdx.x + col];
//
//			}
//			else equal++;
//		}
//		if (less <= (dim*dim + 1) / 2 && greater <= (dim*dim + 1) / 2) break;
//		else if (less>greater) max = maxltguess;
//		else min = mingtguess;
//	}
//	int val = 0;
//	if (less >= (dim*dim + 1) / 2) val = maxltguess;
//	else if (less + equal >= (dim*dim + 1) / 2) val = guess;
//	else val = mingtguess;
//
//	outputImage[y*width + x] = val;
//
//	
//}