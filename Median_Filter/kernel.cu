#include "hip/hip_runtime.h"
#define N 512
#define BLOCK_DIM 32

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
//#include "Filter.h"
#include "Grid.h"
#include "ReadWrite.h"
#include "Filter.h"


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int i = col + row * N;
	if (col < N && row < N) 
		c[i] = a[i] + b[i];
}



int main()
{
	char* name = "Points_[1.0e+08]_Noise_[030]_Normal.bin";
	double t1, t2;
	//Grid grid_s = Grid(4096, 4096);
	Grid grid_p = Grid(100, 100);
	//Grid grid_p_b = Grid(4096, 4096);
	printf("OpenMP threads: %d\n", omp_get_max_threads());
	//t1 = omp_get_wtime();
	printf("Binning file");
	ReadWrite::LoadData_omp(grid_p, name);
	//t2 = omp_get_wtime();
	//printf("Time for omp binning: %12.3f sec, checksum=%d (must be 100000000).\n", t2 - t1, grid_p.Count());
	//t1 = omp_get_wtime();
	//ReadWrite::LoadData_s(grid_s, name);
	//ReadWrite::LoadData_omp_buffer(grid_p_b, name);
	//t2 = omp_get_wtime();
	//printf("Time for serial binning: %12.3f sec, checksum=%d (must be 100000000).\n", t2 - t1, grid_s.Count());
	//grid_s.Print();
	//t1 = 0.0;
	//t2 = 0.0;	
	printf("Writing unfiltered");
	ReadWrite::WriteData(grid_p, "unfiltered.csv");
	
	
	
	//grid_p.Print();
	printf("filtering");
	t1 = omp_get_wtime();
	Filter::m_Filter_extended(grid_p, 3);
	t2 = omp_get_wtime();
	printf("Time for serial Filtering: %12.3f sec.\n", t2 - t1);
	printf("writing filtered\n");
	ReadWrite::WriteData(grid_p, "filtered.csv");
	//grid_p.Print();
	//grid_p_b.Print();
    //const int arraySize = 5;
    //int *a = new int[N*N];
	//int *b = new int[N*N];
	//int *c = new int[N*N];
	//int arraySize = N*N;

	//for (int x = 0; x < N*N; ++x)
	//{
	//	a[x] = 1;
	//	b[x] = 0;
	//	c[x] = 0;
	//}

    // Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
     //   return 1;
   // }
	

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
	dim3 dimGrid((int)ceil(N / (float)dimBlock.x), (int)ceil(N / (float)dimBlock.y));
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<dimGrid, dimBlock>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
