#include "hip/hip_runtime.h"
#ifndef __HIPCC__  
#define __HIPCC__
#endif

#define N 128
#define BLOCK_DIM 4
#define T 32

#define BLOCK_W 32
#define BLOCK_H 32
#define R 9
#define D (R*2+1)
#define D2 D*D
#define TILE_W (BLOCK_W - (2*R))
#define TILE_H (BLOCK_H - (2*R))

//#define TILE_W 12
//#define TILE_H 12

//#define BLOCK_W (TILE_W + (2*R))
//#define BLOCK_H (TILE_H + (2*R))


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

#include "Grid.h"
#include "ReadWrite.h"
#include "Filter.h"


//fix for size 1 larger than block

hipError_t addWithCuda(int *outputImage, const int *inputImage, unsigned int inputSize, unsigned int outputSize);

__global__ void addKernel(int *inputImage, int *outputImage, int width, int height)//int *inputImage, int *outputImage, unsigned int width, unsigned int height)
{
	//int idc, val, min, max, inf, equal, sup, mxinf, minsup, estim;

	//int ib = threadIdx.y;
	//int jb = threadIdx.x;
	//int idx_h = __mul24(ib+R, blockDim.x+2) + jb + 1;
	//int offset = __mul24(blockDim.x, R);

	//int j = __mul24(blockIdx.x, blockDim.x) + jb;
	//int i = __mul24(blockIdx.y, blockDim.y) + ib;
	//if (i < N && j < N)
	//{
	//	/*extern*/ __shared__ int buff[40 * 40];
	//	buff[idx_h] = inputImage[i * N + j];
	//	if (ib < R)
	//	{
	//		buff[idx_h - offset] = 0;//padding
	//	}
	//	else if (ib >= (blockDim.y-R))
	//	{
	//		buff[idx_h + offset] = 0;//padding
	//	}
	//	__syncthreads();
	//	printf("dim[%d][%d] | %-10d : %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d \n", i, j, inputImage[(i) * N + j],
	//		buff[idx_h-offset-1], buff[idx_h-offset], buff[idx_h-offset+1],
	//		buff[idx_h-1], buff[idx_h], buff[idx_h+1], 
	//		buff[idx_h+offset-1], buff[idx_h+offset], buff[idx_h+offset]);
	//	min = max = buff[ib * blockDim.x + jb];

	//	/*for (idc = 0; idc < 2 * R + 1; ++idc)
	//	{
	//	val = buff[__mul24(ib + idc, blockDim.x) + jb];
	//	if (val < min) min = val;
	//	if (val > max) max = val;
	//	}

	//	while (1)
	//	{
	//	estim = (min + max) / 2;
	//	inf = sup = equal = 0;
	//	mxinf = min;
	//	minsup = max;
	//	for (idc = 0; idc < 2 * R + 1; ++idc)
	//	{
	//	val = buff[__mul24(ib + idc, blockDim.x) + jb];
	//	if (val < estim)
	//	{
	//	inf++;
	//	if (val > mxinf) mxinf = val;
	//	}
	//	else if (val > estim)
	//	{
	//	sup++;
	//	if (val < minsup) minsup = val;
	//	}
	//	else equal++;
	//	}
	//	if ((inf <= (R + 1)) && (sup <= (R + 1))) break;
	//	else if (inf > sup) max = mxinf;
	//	else min = minsup;
	//	}
	//	if (inf >= R + 1) val = mxinf;
	//	else if (inf + equal >= R + 1) val = estim;
	//	else val = minsup;*/

	//	outputImage[__mul24(j, N) + i] = 0;//val;
	//}

//	__shared__ int smem[16 * 16];
//
//	int x = blockIdx.x*TILE_W + threadIdx.x - R;
//
//	int y = blockIdx.y*TILE_H + threadIdx.y - R;
//
//	//clamp to edge of image
////	if (blockIdx.x*blockDim.x + threadIdx.x < N + R && blockIdx.y*blockDim.y + threadIdx.y < N + R)
//	//{
//		x = max(0, x);
//
//		x = min(x, width - 1);
//
//		y = max(y, 0);
//
//		y = min(y, height - 1);
//
//
//
//		unsigned int index = y*width + x;
//
//		unsigned int bindex = threadIdx.y * blockDim.y + threadIdx.x;
//
//
//
//		//each thread copies its pixel of the block to shared memory
//
//		smem[bindex] = inputImage[index];
//
//		__syncthreads();
//
//		if ((threadIdx.x >= R) && (threadIdx.x < (BLOCK_W - R)) &&
//			(threadIdx.y >= R) && (threadIdx.y < (BLOCK_H - R)))
//		{
//			
//			min = max = smem[ib * blockDim.x + jb];
//
//				for (idc = 0; idc < 2 * R + 1; ++idc)
//				{
//					val = smem[__mul24(ib + idc, blockDim.x) + jb];
//				if (val < min) min = val;
//				if (val > max) max = val;
//				}
//
//				while (1)
//				{
//				estim = (min + max) / 2;
//				inf = sup = equal = 0;
//				mxinf = min;
//				minsup = max;
//				for (idc = 0; idc < 2 * R + 1; ++idc)
//				{
//					val = smem[__mul24(ib + idc, blockDim.x) + jb];
//				if (val < estim)
//				{
//				inf++;
//				if (val > mxinf) mxinf = val;
//				}
//				else if (val > estim)
//				{
//				sup++;
//				if (val < minsup) minsup = val;
//				}
//				else equal++;
//				}
//				if ((inf <= (R + 1)) && (sup <= (R + 1))) break;
//				else if (inf > sup) max = mxinf;
//				else min = minsup;
//				}
//				if (inf >= R + 1) val = mxinf;
//				else if (inf + equal >= R + 1) val = estim;
//				else val = minsup;
//
//			/*int      val, i, less, greater, equal,  min, max, guess, maxltguess, mingtguess;
//
//			min = max = smem[bindex];
//			for (i =-1; i<=1; i++) {
//			val = smem[__mul24(threadIdx.y + i, blockDim.x) + threadIdx.x];
//			if (val<min) min = val;
//			if (val> max) max = val;
//			}
//
//			while (1) {
//				guess = (min + max) / 2;
//				less = 0; greater = 0; equal = 0;
//				maxltguess = min;
//				mingtguess = max;
//				for (i = -1; i<=1; i++) {
//					val = smem[__mul24(threadIdx.y + i, blockDim.x) + threadIdx.x];
//					if (val<guess) {
//						less++;
//						if (val>maxltguess) maxltguess = val;
//					}
//					else if (val>guess) {
//						greater++;
//					if (val<mingtguess) mingtguess = val;
//					}
//					else equal++;
//				}
//				if (less <= (R + 1) && greater <= (R + 1)) break;
//				else if (less>greater) max = maxltguess;
//				else min = mingtguess;
//			}
//			if (less >= (R + 1)) val = maxltguess;
//			else if (less + equal >= (R + 1)) val = guess;
//			else val = mingtguess;*/
//			/*printf("dim[%d][%d] | %-10d : %-10d, %-10d, %-10d, %-10d, %-10d, %-10d, %-10d, %-10d, %-10d \n", y, x, inputImage[index],
//				smem[bindex - blockDim.x - 1], smem[bindex - blockDim.x], smem[bindex - blockDim.x + 1],
//				smem[bindex - 1], smem[bindex], smem[bindex + 1],
//				smem[bindex + blockDim.x - 1], smem[bindex + blockDim.x], smem[bindex + blockDim.x + 1]);*/
//			//outputImage[index] =  smem[bindex];
//			outputImage[index] = val;
//
//		}
	//}

	 //extern __shared__ int data[];
	__shared__ int data[(R * 2 + 1) * (T + 2 * R)];
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int d_col = col + R;
	int d_row = blockIdx.y;

	if (row < N && col < N)
	{

		for (int r = -R; r <= R; ++r)
		{
			//col += r;
			int i_row = row + r;
			//col = max(0, col);

			//col = min(col, N - 1);

			i_row = max(i_row, 0);

			i_row = min(i_row, N - 1);
			//data[1028 + col] = 0; //inputImage[row*N + col];
			data[(R + r) * (T + 2 * R) + d_col] = inputImage[i_row*N + col];
			if (col == 0)
			{
				for (int i = 0; i <= R;++i)
					data[(R + r) * (T + 2 * R) + col+i] = inputImage[i_row*N + col];
			}
			if (col == N - 1 || col == blockDim.x)
			{
				for (int i = 0; i <= R; ++i)
					data[(R + r) * (T + 2 * R) + d_col + i] = inputImage[i_row*N + col];
			}
		}
		__syncthreads();
		/*printf("dim : %d | [%d][%d] | %-10d : %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d, %-9d \n", blockDim.y, d_row, col, inputImage[(row) * N + col], 
			data[(0) * 514 + d_col - 1], data[(0) * 514 + d_col], data[(0) * 514 + d_col + 1],
			data[(1) * 514 + d_col - 1], data[(1) * 514 + d_col], data[(1) * 514 + d_col + 1],
			data[(2) * 514 + d_col - 1], data[(2) * 514 + d_col], data[(2) * 514 + d_col + 1]);*/
		//printf("dim : %d | [%d][%d] | %-10d : %-9d, %-9d, %-9d\n", blockDim.y, d_row, N, inputImage[(row) * N + col], data[(0) * 514 + N], data[(1)* 514 + N], data[(2) * 514 + N]);
		//int k = 0;
		//int *window = new int[D*D];
		//for (int dy = -R; dy <= R; ++dy)
		//{
		//	for (int dx = -R; dx <= R; ++dx)
		//	{
		//		window[k++] = data[(R + dy) * (T + 2 * R) + d_col + dx];
		//	}
		//}
		//for (int j = 0; j < (D*D+1)/2; ++j)
		//{
		//	//   find position of minimum element
		//	int min = j;
		//	for (int l = j + 1; l < (D*D); ++l)
		//		if (window[l] < window[min])
		//			min = l;
		//	const int temp = window[j];
		//	window[j] = window[min];
		//	window[min] = temp;
		//}
		/*int      val, i, less, greater, equal,  min, max, guess, maxltguess, mingtguess;
		
		min = max = data[__mul24(threadIdx.y + R, blockDim.x + 2 * R) + threadIdx.x + R];
		for (i =0; i<D; i++) {
			val = data[__mul24(threadIdx.y + i, blockDim.x) + threadIdx.x];
			if (val<min) min = val;
			if (val> max) max = val;
		}
		while (1) {
			guess = (min + max) / 2;
			less = 0; greater = 0; equal = 0;
			maxltguess = min;
			mingtguess = max;
			for (i = 0; i<D; i++) {
				val = data[__mul24(threadIdx.y + i, blockDim.x) + threadIdx.x];
				if (val<guess) {
					less++;
					if (val>maxltguess) maxltguess = val;
				}
				else if (val>guess) {
					greater++;
				if (val<mingtguess) mingtguess = val;
				}
				else equal++;
			}
			if (less <= (R + 1) && greater <= (R + 1)) break;
			else if (less>greater) max = maxltguess;
			else min = mingtguess;
		}
		if (less >= (R + 1)) val = maxltguess;
		else if (less + equal >= (R + 1)) val = guess;
		else val = mingtguess;*/

		//outputImage[(row)* N + col] = val;
		//outputImage[(row)* N + col] = data[(R)* (T + 2 * R) + d_col/*(1) * 514 + d_col*/];
		outputImage[(row)* N + col] = 1;// window[(D*D) / 2];
	}
}

__global__ void median_Kernel_3x3(int *inputImage, int *outputImage, int width, int height)//int *inputImage, int *outputImage, unsigned int width, unsigned int height)
{
__shared__ int smem[(BLOCK_DIM + 2 * R)][(BLOCK_DIM + 2 * R)];
int x_s = threadIdx.x + R;
int y_s = threadIdx.y + R;

int x_a = blockIdx.x*blockDim.x + threadIdx.x + R;
int y_a = blockIdx.y*blockDim.y + threadIdx.y + R;

int x = blockIdx.x*blockDim.x + threadIdx.x;
int y = blockIdx.y*blockDim.y + threadIdx.y;
if (x_a < N + 2 * R && y_a < N + 2 * R)
{
	smem[y_s][x_s] = inputImage[y_a*(N + 2 * R) + x_a];
	if (threadIdx.x == 0 || threadIdx.x == BLOCK_DIM - 1)
	{
		for (int i = 0; i < R; ++i)
		{
			smem[threadIdx.y][threadIdx.x + i] = inputImage[y*(N + 2 * R) + x + i];
		}
	}
	if (threadIdx.y == 0 || threadIdx.y == BLOCK_DIM - 1)
	{
		for (int i = 0; i < R; ++i)
		{
			smem[threadIdx.y + i][threadIdx.x] = inputImage[(y + i)*(N + 2 * R) + x];
		}
	}

	__syncthreads();
	if (x < N && y < N)
	{
		/*printf("dim[%d][%d] | %-10d : \n%-10d, %-10d, %-10d, \n%-10d, %-10d, %-10d, \n%-10d, %-10d, %-10d \n", y, x, inputImage[y_a*(N + 2 * R) + x_a],
			smem[y_a - 1][x_a - 1], smem[y_a - 1][x_a], smem[y_a - 1][x_a + 1],
			smem[y_a][x_a - 1], smem[y_a][x_a], smem[y_a][x_a + 1],
			smem[y_a + 1][x_a - 1], smem[y_a+1][x_a], smem[y_a + 1][x_a + 1]);*/
		outputImage[y*width + x] = smem[y_s][x_s];
	}
}
//how did you change your seat
}

__global__ void median_Kernel_5x5(int *inputImage, int *outputImage, int width, int height)//int *inputImage, int *outputImage, unsigned int width, unsigned int height)
{
	__shared__ int smem[(BLOCK_DIM + 2 * R)][(BLOCK_DIM + 2 * R)];
	int x_s = threadIdx.x + R;
	int y_s = threadIdx.y + R;

	int x_a = blockIdx.x*blockDim.x + threadIdx.x + R;
	int y_a = blockIdx.y*blockDim.y + threadIdx.y + R;

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if (x < N && y < N)
	{
		smem[y_s][x_s] = inputImage[y*(N)+x];
		if (threadIdx.x == 0 && threadIdx.y == 0 && x != N - 1 && y != N - 1)
		{
			for (int j = 0; j < R; ++j)
				for (int i = 0; i < R; ++i)
					smem[threadIdx.y + j][threadIdx.x + i] = inputImage[y*(N)+x];
		}
		else if (threadIdx.x == BLOCK_DIM - 1 && threadIdx.y == BLOCK_DIM - 1)
		{
			for (int j = 0; j < R; ++j)
				for (int i = 0; i < R; ++i)
					smem[y_s + j + 1][x_s + i + 1] = inputImage[y*(N)+x];

		}
		else if (threadIdx.x == 0 && threadIdx.y == BLOCK_DIM - 1)
		{
			for (int j = 0; j < R; ++j)
				for (int i = 0; i < R; ++i)
					smem[y_s + j + 1][threadIdx.x + i] = inputImage[y*(N)+x];

		}
		else if (threadIdx.x == BLOCK_DIM - 1 && threadIdx.y == 0)
		{
			for (int j = 0; j < R; ++j)
				for (int i = 0; i < R; ++i)
					smem[threadIdx.y + j][x_s + i + 1] = inputImage[y*(N)+x];

		}

		if (threadIdx.x == 0 || x != N - 1)
		{
			for (int i = 0; i < R; ++i)
			{
				smem[y_s][threadIdx.x + i] = inputImage[y*(N)+x];
			}
		}
		else if (threadIdx.x == BLOCK_DIM - 1)
		{
			for (int i = 1; i <= R; ++i)
			{
				smem[y_s][x_s + i] = inputImage[y*(N)+x];
			}
		}
		if (threadIdx.y == 0)
		{
			for (int i = 0; i < R; ++i)
			{
				smem[threadIdx.y + i][x_s] = inputImage[y*(N)+x];
			}
		}
		else if (threadIdx.y == BLOCK_DIM - 1)
		{
			for (int i = 1; i <= R; ++i)
			{
				smem[y_s + i][x_s] = inputImage[y*(N)+x];
			}
		}

		__syncthreads();

		const int dim = D;
		int window[dim*dim];
		for (int j = -R; j <= R; ++j)
			for (int i = -R; i <= R; ++i)
				window[(R - j)*dim + R - i] = smem[y_s + j][x_s + i];
		for (int j = 0; j < (dim*dim + 1) / 2; ++j)
		{
			//   Find position of minimum element
			int min = j;
			for (int l = j + 1; l < dim*dim; ++l)
				if (window[l] < window[min])
					min = l;
			//   Put found minimum element in its place
			const int temp = window[j];
			window[j] = window[min];
			window[min] = temp;
		}
		/*printf("dim[%d][%d] | %-10d : \n\t\t\t%-10d, %-10d, %-10d, \n\t\t\t%-10d, %-10d, %-10d, \n\t\t\t%-10d, %-10d, %-10d \n", y, x, inputImage[y*(N) + x],
		smem[y_s - 1][x_s - 1], smem[y_s - 1][x_s], smem[y_s - 1][x_s + 1],
		smem[y_s][x_s - 1], smem[y_s][x_s], smem[y_s][x_s + 1],
		smem[y_s + 1][x_s - 1], smem[y_s + 1][x_s], smem[y_s + 1][x_s + 1]);*/

		printf("dim[%d][%d] | %-10d : \n%-10d, %-10d, %-10d, %-10d, %-10d, \n%-10d, %-10d, %-10d, %-10d, %-10d, \n%-10d, %-10d, %-10d, %-10d, %-10d, \n%-10d, %-10d, %-10d, %-10d, %-10d, \n%-10d, %-10d, %-10d, %-10d, %-10d,\n", y, x, inputImage[y*(N)+x],
			smem[y_s - 2][x_s - 2], smem[y_s - 2][x_s - 1], smem[y_s - 2][x_s], smem[y_s - 2][x_s + 1], smem[y_s - 2][x_s + 2],
			smem[y_s - 1][x_s - 2], smem[y_s - 1][x_s - 1], smem[y_s - 1][x_s], smem[y_s - 1][x_s + 1], smem[y_s - 1][x_s + 2],

			smem[y_s][x_s - 2], smem[y_s][x_s - 1], smem[y_s][x_s], smem[y_s][x_s + 1], smem[y_s][x_s + 2],

			smem[y_s + 1][x_s - 2], smem[y_s + 1][x_s - 1], smem[y_s + 1][x_s], smem[y_s + 1][x_s + 1], smem[y_s + 1][x_s + 2],
			smem[y_s + 2][x_s - 2], smem[y_s + 2][x_s - 1], smem[y_s + 2][x_s], smem[y_s + 2][x_s + 1], smem[y_s + 2][x_s + 2]
			);

		outputImage[y*width + x] = window[dim*dim / 2];
	}
	//how did you change your seat
}

__global__ void median_Kernel_21x21(int *inputImage, int *outputImage, int width, int height)//int *inputImage, int *outputImage, unsigned int width, unsigned int height)
{

	__shared__ int smem[BLOCK_W * BLOCK_H];

	int x = blockIdx.x*TILE_W + threadIdx.x - R;

	int y = blockIdx.y*TILE_H + threadIdx.y - R;

	int a = blockIdx.x*blockDim.x + threadIdx.x;
	int b = blockIdx.y*blockDim.y + threadIdx.y;

	//clamp to edge of image
	if (x < N + 2 && y < N + 2)
	{
		x = max(0, x);

		x = min(x, width - 1);

		y = max(y, 0);

		y = min(y, height - 1);



		unsigned int index = y*width + x;

		unsigned int bindex = threadIdx.y * blockDim.y + threadIdx.x;



		//each thread copies its pixel of the block to shared memory

		smem[bindex] = inputImage[index];

		__syncthreads();
		if ((threadIdx.x >= R) && (threadIdx.x < BLOCK_W - R) && (threadIdx.y >= R) && (threadIdx.y < BLOCK_W - R))

		{

			if (blockIdx.x*TILE_W + threadIdx.x < N + 2 && blockIdx.y*TILE_H + threadIdx.y < N + 2)
			{
				/*printf("[%d][%d] dim[%d][%d] | %-10d : \n%-10d, %-10d, %-10d, %-10d, %-10d, \n%-10d, %-10d, %-10d, %-10d, %-10d, \n%-10d, %-10d, %-10d, %-10d, %-10d, \n%-10d, %-10d, %-10d, %-10d, %-10d, \n%-10d, %-10d, %-10d, %-10d, %-10d,\n", b, a, y, x, inputImage[y*(N)+x],
					smem[bindex - 2 - BLOCK_W - BLOCK_W], smem[bindex - 1 - BLOCK_W - BLOCK_W], smem[bindex - BLOCK_W - BLOCK_W], smem[bindex + 1 - BLOCK_W - BLOCK_W], smem[bindex + 2 - BLOCK_W - BLOCK_W],
					smem[bindex - 2 - BLOCK_W], smem[bindex - 1 - BLOCK_W], smem[bindex - BLOCK_W], smem[bindex + 1 - BLOCK_W], smem[bindex + 2 - BLOCK_W],

					smem[bindex - 2], smem[bindex - 1], smem[bindex], smem[bindex + 1], smem[bindex + 2],

					smem[bindex - 2 + BLOCK_W], smem[bindex - 1 + BLOCK_W], smem[bindex + BLOCK_W], smem[bindex + 1 + BLOCK_W], smem[bindex + 2 + BLOCK_W],
					smem[bindex - 2 + BLOCK_W + BLOCK_W], smem[bindex - 1 + BLOCK_W + BLOCK_W], smem[bindex + BLOCK_W + BLOCK_W], smem[bindex + 1 + BLOCK_W + BLOCK_W], smem[bindex + 2 + BLOCK_W + BLOCK_W]
					);*/

				const int dim = D;
				int window[dim*dim];
				for (int j = -R; j <= R; ++j)
					for (int i = -R; i <= R; ++i)
						window[(R - j)*dim + R - i] = smem[bindex + j*BLOCK_W + i];
				for (int j = 0; j < (dim*dim + 1) / 2; ++j)
				{
					//   Find position of minimum element
					int min = j;
					for (int l = j + 1; l < dim*dim; ++l)
						if (window[l] < window[min])
							min = l;
					//   Put found minimum element in its place
					const int temp = window[j];
					window[j] = window[min];
					window[min] = temp;
				}
				outputImage[y*width + x] = window[dim*dim / 2];
				//outputImage[index] = smem[bindex];
			}

			//outputImage[index] = smem[bindex];
		}
	}
}

int main()
{
	char* name = "Points_[1.0e+08]_Noise_[030]_Normal.bin";
	double t1, t2;
	Grid grid_p = Grid(N, N);
	printf("CONFIGURATION\n------------------\n");
	printf("Bin dimensions : %d - %d\n", N, N);
	printf("Filter dimensions : %d - %d\n", D, D);
	printf("OpenMP threads: %d\n", omp_get_max_threads());
	printf("Binning file\n");
	ReadWrite::LoadData_omp(grid_p, name);
	printf("original grid : %d\n", grid_p.Count());
	//grid_p.Print();
	printf("padding grid\n");
	t1 = omp_get_wtime();
	Grid padded = grid_p.Pad(R);
	t2 = omp_get_wtime();
	printf("Time for padding: %12.3f sec.\n", t2 - t1);
	Grid input = grid_p;
	//padded.Print();
	//grid_p.Print();
	t1 = omp_get_wtime();
	//Filter::m_Filter_half(padded, grid_p, D);
	t2 = omp_get_wtime();
	//printf("Time for half Filtering: %12.3f sec.\n", t2 - t1);
	 
    // Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	Grid cuda = Grid(N, N);
	printf("filtered serial grid : %d\n", grid_p.Count());
	//grid_p.Print();
	//padded.Print();
	//grid_p.Print();
	 
	//cuda.set(0);
	hipError_t cudaStatus = addWithCuda(cuda.grid, input.grid, input.cols*input.rows, cuda.rows*cuda.cols);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	printf("filtered cuda grid : %d\n", cuda.Count());
	//cuda.Print();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *outputImage, const int *inputImage, unsigned int inputSize, unsigned int outputSize)
{
	const int size = 32;
	int a[size*size], b[size*size], c[size*size];
	for (int j = 0; j < size*size; ++j)
	{
			a[j] = 1;
			b[j] = 1;
			c[j] = -1;
			if (j%size == 0)
				c[j] = 0;
			
	}

	/*for (int j = 0; j < size; ++j)
	{
		printf("\n");
		for (int i = 0; i < size; ++i)
			printf("%-3d ", c[j * size + i]);
	}
	printf("\n");*/
    int *dev_i = 0;
    int *dev_o = 0;
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_i, inputSize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	cudaStatus = hipMalloc((void**)&dev_o, outputSize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	/*cudaStatus = hipMalloc((void**)&dev_c, 64 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_a, 64 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_b, 64 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}*/

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_i, inputImage, inputSize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	
	/*cudaStatus = hipMemcpy(dev_a, a, 64 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_b, b, 64 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}*/

	dim3 grid((N + TILE_W - 1) / TILE_W, (N + TILE_H - 1) / TILE_W);
	dim3 threads(BLOCK_H, BLOCK_W);
	//dim3 threads(T, 1);
	//dim3 threads(BLOCK_DIM, BLOCK_DIM);
	//dim3 grid((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);
	//dim3 grid((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);
	//dim3 grid(1, 1);
	hipEventRecord(start);
	//double t1, t2;
	//t1 = omp_get_wtime();
    // Launch a kernel on the GPU with one thread for each element.
	//if (R == 1)
	//	median_Kernel_3x3 << <grid, threads/*, 128*128*sizeof(int)*/ >> >(dev_i, dev_o, N, N);
	//if (R == 2)
	//	median_Kernel_5x5 << <grid, threads/*, 128*128*sizeof(int)*/ >> >(dev_i, dev_o, N, N);
	//
	//if (R == 10)
		median_Kernel_21x21 << <grid, threads/*, 128*128*sizeof(int)*/ >> >(dev_i, dev_o, N, N);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("cudatime = %12.12f ms\n", milliseconds);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	//t2 = omp_get_wtime();
	//printf("Time for cuda Filtering: %12.3f sec.\n", t2 - t1);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(outputImage, dev_o, outputSize * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	
	/*cudaStatus = hipMemcpy(c, dev_c, 64 * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }*/

	
Error:
    hipFree(dev_i);
    hipFree(dev_o);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	/*for (int j = 0; j < size; ++j)
	{
		printf("\n");
		for (int i = 0; i < size; ++i)
			printf("%-3d ", c[j * size + i]);
	}*/
    return cudaStatus;
}
