#include "hip/hip_runtime.h"
#ifndef __HIPCC__  
#define __HIPCC__
#endif

#define N 6
#define BLOCK_DIM 8

#define TILE_W 6
#define TILE_H 6
#define R 1
#define D (R*2+1)
#define BLOCK_W (TILE_W + (2*R))
#define BLOCK_H (TILE_H + (2*R))


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

#include "Grid.h"
#include "ReadWrite.h"
#include "Filter.h"


hipError_t addWithCuda(int *outputImage, const int *inputImage, unsigned int paddedsize, unsigned int gridSize);

__global__ void addKernel(int *inputImage, int *outputImage, unsigned int width, unsigned int height)
{
	__shared__ int smem[BLOCK_W*BLOCK_H];

	int x = blockIdx.x*TILE_W + threadIdx.x;
	int y = blockIdx.y*TILE_H + threadIdx.y;
	//		if (x < 5 && y < 5)
	//		{
	//# if __CUDA_ARCH__>=200
	//	printf("%d \n", inputImage[x ]);
	//#endif
//}		
	//printf("x : %d, y : %d \n", x, y);
		x = max(0, x);
		x = min(x, width - 1);
		y = max(y, 0);
		y = min(y, height - 1);
		//printf("x : %d, y : %d \n", x, y);

		unsigned int index = y * width + x;
		unsigned int bindex = threadIdx.y*blockDim.y + threadIdx.x;
		//printf("current [%d][%d] : %d\n", threadIdx.y, threadIdx.x, inputImage[index]);
		smem[bindex] = inputImage[index];
		__syncthreads();

		if ((threadIdx.x >= R) && (threadIdx.x < (BLOCK_W - R)) &&
			(threadIdx.y >= R) && (threadIdx.y < (BLOCK_H - R)))
		{
			printf("current [%d][%d] : %d\n", threadIdx.y, threadIdx.x, smem[bindex]);
			float sum = 0;
			int k = 0;
			int *window = new int[9];
			for (int dy = -R; dy <= R; ++dy)
			{
				for (int dx = -R; dx <= R; ++dx)
				{
					window[k++] = smem[bindex + (dy*blockDim.x) + dx];
				}
			}
			for (int j = 0; j < 5; ++j)
			{
				//   Find position of minimum element
				int min = j;
				for (int l = j + 1; l < 9; ++l)
					if (window[l] < window[min])
						min = l;
				//   Put found minimum element in its place
				const int temp = window[j];
				window[j] = window[min];
				window[min] = temp;
			}

			//outputImage[index] = window[4];
			outputImage[index] = smem[bindex];

		}
}




int main()
{
	int size = 3;
	char* name = "Points_[1.0e+08]_Noise_[030]_Normal.bin";
	double t1, t2;
	//Grid grid_s = Grid(512, 512);
	Grid grid_p = Grid(6, 6);
	//Grid grid_p_b = Grid(4096, 4096);
	printf("OpenMP threads: %d\n", omp_get_max_threads());
	//t1 = omp_get_wtime();
	printf("Binning file\n");
	ReadWrite::LoadData_omp(grid_p, name);
	//grid_p.Print();
	printf("padding grid\n");
	int radius = (int)(size - 1) / 2;
	Grid padded = grid_p.Pad(radius);
	padded.Print();
	//t1 = omp_get_wtime();
	//Filter::m_Filter_fullsort(padded, grid_p, radius);
	//t2 = omp_get_wtime();
	//printf("Time for full Filtering: %12.3f sec.\n", t2 - t1);
	//printf("filtered grid\n");
	//grid_p.Print();
	t1 = omp_get_wtime();
	//Filter::m_Filter_half(padded, grid_p, size);
	t2 = omp_get_wtime();
	printf("Time for half Filtering: %12.3f sec.\n", t2 - t1);
	//grid_p.Print();
	//printf("grid[0][0] = %d\n", grid_p.grid[0]);
	//t2 = omp_get_wtime();
	//printf("Time for omp binning: %12.3f sec, checksum=%d (must be 100000000).\n", t2 - t1, grid_p.Count());
	//t1 = omp_get_wtime();
	//ReadWrite::LoadData_omp(grid_s, name);
	//ReadWrite::LoadData_s(grid_s, name);
	//ReadWrite::LoadData_omp_buffer(grid_p_b, name);
	//t2 = omp_get_wtime();
	//printf("Time for serial binning: %12.3f sec, checksum=%d (must be 100000000).\n", t2 - t1, grid_s.Count());
	//grid_s.Print();
	//t1 = 0.0;
	//t2 = 0.0;	
	//printf("Writing unfiltered");
	//ReadWrite::WriteData(grid_p, "unfiltered.csv");
	
	
	
	/*grid_p.Print();
	printf("grid_count : %d\n", grid_p.Count());
	printf("filtering");
	t1 = omp_get_wtime();
	Filter::m_Filter_extended(grid_p, radius);
	t2 = omp_get_wtime();
	printf("Time for serial Filtering: %12.3f sec.\n", t2 - t1);
	printf("writing filtered\n");*/
	//ReadWrite::WriteData(grid_p, "filtered.csv");
	//grid_p.Print();
	//grid_p_b.Print();
    //const int arraySize = 5;
    /*int *a = new int[N*N];
	int *b = new int[N*N];
	int *c = new int[N*N];
	unsigned int arraySize = N*N;

	for (int x = 0; x < N*N; ++x)
	{
		a[x] = 1;
		b[x] = 0;
		c[x] = 0;
	}*/

    // Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	printf("filtered grid\n");
	grid_p.Print();
	//padded.Print();
	hipError_t cudaStatus = addWithCuda(grid_p.grid, padded.grid, padded.cols*padded.rows, grid_p.rows*grid_p.cols);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	grid_p.Print();
	//printf("c[0][0] = %d\n", c[0]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *outputImage, const int *inputImage, unsigned int paddedsize, unsigned int gridSize)
{
    int *dev_i = 0;
    int *dev_o = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_o, gridSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_i, paddedsize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_i, inputImage, paddedsize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
	dim3 dimGrid((N + dimBlock.x)/dimBlock.x, (N + dimBlock.y)/dimBlock.y);
	hipEventRecord(start);
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<dimGrid, dimBlock>>>(dev_i, dev_o, 8, 8);
	hipEventRecord(stop);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(outputImage, dev_o, gridSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("cudatime = %12.12f\n", milliseconds/1000);
Error:
    hipFree(dev_i);
    hipFree(dev_o);
    
    return cudaStatus;
}
