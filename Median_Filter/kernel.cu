#include "hip/hip_runtime.h"
#define N 512
#define BLOCK_DIM 32

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
//#include "Filter.h"
#include "Grid.h"
#include "ReadWrite.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int i = col + row * N;
	if (col < N && row < N) 
		c[i] = a[i] + b[i];
}

int main()
{
	double t1, t2;
	Grid grid_s = Grid(5, 5);
	Grid grid_p = Grid(5, 5);

	t1 = omp_get_wtime();
	ReadWrite::LoadData_s(grid_s);
	t2 = omp_get_wtime();
	printf("Time for serial binning: %12.3f sec, checksum=%d (must be 100000000).\n", t2 - t1, grid_s.Count());
	grid_s.Print();
	t1 = 0.0;
	t2 = 0.0;
	t1 = omp_get_wtime();
	ReadWrite::LoadData_omp(grid_p);
	t2 = omp_get_wtime();
	printf("Time for serial omp: %12.3f sec, checksum=%d (must be 100000000).\n", t2 - t1, grid_p.Count());
	grid_p.Print();
    //const int arraySize = 5;
    //int *a = new int[N*N];
	//int *b = new int[N*N];
	//int *c = new int[N*N];
	//int arraySize = N*N;

	//for (int x = 0; x < N*N; ++x)
	//{
	//	a[x] = 1;
	//	b[x] = 0;
	//	c[x] = 0;
	//}

    // Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
     //   return 1;
   // }
	

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
	dim3 dimGrid((int)ceil(N / (float)dimBlock.x), (int)ceil(N / (float)dimBlock.y));
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<dimGrid, dimBlock>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
